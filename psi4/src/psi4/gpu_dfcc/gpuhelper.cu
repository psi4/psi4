/*
 *@BEGIN LICENSE
 *
 * GPU-accelerated density-fitted coupled-cluster, a plugin to:
 *
 * PSI4: an ab initio quantum chemistry software package
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 *
 *@END LICENSE
 */

// TODO: interleaved dgemm seems to be broken
bool interleaved_dgemm = true;

#include<psi4/libplugin/plugin.h>
#include<psi4/psi4-dec.h>
#include"psi4/liboptions/liboptions.h"
#include<psi4/libqt/qt.h>
#include<psi4/libtrans/integraltransform.h>
#include<psi4/libtrans/mospace.h>
#include<psi4/libmints/matrix.h>
#include<psi4/libmints/vector.h>
#include<psi4/libiwl/iwl.h>
#include<psi4/libpsio/psio.hpp>
#include<psi4/libparallel/process.h>
#include"blas.h"

#include"gpuhelper.h"
#include"gpuonly.h"
#include<omp.h>
using namespace psi;
using namespace std;

namespace psi{namespace fnocc{

void GPUHelper::Check_CUDA_Error(FILE*fp,const char *message){
  hipError_t error = hipGetLastError();
  if (error!=hipSuccess) {
     fprintf(fp,"\n  ERROR: %s: %s\n\n", message, hipGetErrorString(error) );
     fflush(fp);
     exit(-1);
  }
}

/*===================================================================

  initialize cublas and get device properties

===================================================================*/
void GPUHelper::CudaInitGPU(Options&options){

  max_mapped_memory=0;
  num_gpus=gpumemory=extraroom=0;
  int n;
  size_t free;
  size_t total;
  hipGetDeviceCount(&n);
  num_gpus = n;
  num_cpus=0;
  if (options["NUM_GPUS"].has_changed())
     num_gpus = options.get_int("NUM_GPUS");

  if (num_gpus>0){
     cublasInit();
     struct hipDeviceProp_t cudaProp;
     int gpu_id;
     hipGetDevice(&gpu_id);
     hipGetDeviceProperties( &cudaProp,gpu_id );
     outfile->Printf(
       "\n  _________________________________________________________\n");
     outfile->Printf("  CUDA device properties:\n");
     outfile->Printf("  name:                 %20s\n",cudaProp.name);
     outfile->Printf("  major version:        %20d\n",cudaProp.major);
     outfile->Printf("  minor version:        %20d\n",cudaProp.minor);
     outfile->Printf("  canMapHostMemory:     %20d\n",cudaProp.canMapHostMemory);
     outfile->Printf("  totalGlobalMem:       %20lu mb\n",
       cudaProp.totalGlobalMem/(1024*1024));
     outfile->Printf("  sharedMemPerBlock:    %20lu\n",cudaProp.sharedMemPerBlock);
     outfile->Printf("  clockRate:            %20.3f ghz\n",
       cudaProp.clockRate/1.0e6);
     outfile->Printf("  regsPerBlock:         %20d\n",cudaProp.regsPerBlock);
     outfile->Printf("  warpSize:             %20d\n",cudaProp.warpSize);
     outfile->Printf("  maxThreadsPerBlock:   %20d\n",cudaProp.maxThreadsPerBlock);
     outfile->Printf(
       "  _________________________________________________________\n\n");
     //fflush(outfile);

     //gpumemory = cudaProp.totalGlobalMem;
     
     
     hipMemGetInfo(&free,&total);
     gpumemory = free; 
     extraroom = 200L*1024L*1024L;
     hipDeviceReset();

     // default memory for mapped cpu memory is the sum of all gpu memory
     max_mapped_memory = (num_gpus+num_cpus) * (gpumemory-extraroom);
     if (options["MAX_MAPPED_MEMORY"].has_changed()){
        long int temp_mem = options.get_int("MAX_MAPPED_MEMORY");
        temp_mem *= 1024L*1024L;
        if (temp_mem<max_mapped_memory)
           max_mapped_memory = temp_mem;
     }
     max_mapped_memory_per_thread = max_mapped_memory/(num_gpus+num_cpus);

     outfile->Printf("\n");
     outfile->Printf("  allocating gpu memory...");
     //fflush(outfile);
     tmp = (double**)malloc(num_gpus*sizeof(double*));   
     gpubuffer = (double**)malloc(num_gpus*sizeof(double*));
     #pragma omp parallel for schedule (static) num_threads(num_gpus)
     for (long int i=0; i<num_gpus; i++){
         long int thread = 0;
         #ifdef _OPENMP
           thread = omp_get_thread_num();
         #endif
         hipSetDevice(thread);
         Check_CUDA_Error(stdout,"hipSetDevice");
         hipHostMalloc((void**)&tmp[thread],max_mapped_memory_per_thread);  
         //tmp[thread] = (double*)malloc(max_mapped_memory_per_thread*sizeof(double));
         Check_CUDA_Error(stdout,"cpu tmp");
         //hipMemGetInfo(&free,&total);
         hipMalloc((void**)&gpubuffer[thread],gpumemory-extraroom);
    //     hipMalloc((void**)&gpubuffer[thread],gpumemory-extraroom);   
         Check_CUDA_Error(stdout,"gpu memory");

     }
     // thread-safe tiling info: TODO: these are never free'd at the end
     myntilesM = (long int*)malloc(num_gpus*sizeof(long int));
     myntilesN = (long int*)malloc(num_gpus*sizeof(long int));
     myntilesK = (long int*)malloc(num_gpus*sizeof(long int));
     mytilesizeM = (long int*)malloc(num_gpus*sizeof(long int));
     mytilesizeN = (long int*)malloc(num_gpus*sizeof(long int));
     mytilesizeK = (long int*)malloc(num_gpus*sizeof(long int));
     mylasttileM = (long int*)malloc(num_gpus*sizeof(long int));
     mylasttileN = (long int*)malloc(num_gpus*sizeof(long int));
     mylasttileK = (long int*)malloc(num_gpus*sizeof(long int));
     mytilesizesM = (long int**)malloc(num_gpus*sizeof(long int*));
     mytilesizesN = (long int**)malloc(num_gpus*sizeof(long int*));
     mytilesizesK = (long int**)malloc(num_gpus*sizeof(long int*));

     //fflush(outfile);

     // some cpu memory for cores to use when stealing gpu work 
     //cpuarray = (double**)malloc(num_cpus*sizeof(double*));
     //for (long int i=0; i<num_cpus; i++){
     //    // TODO: need to be more intelligent about this...
     //    cpuarray[i] = (double*)malloc(3*max_mapped_memory_per_thread+20*max_mapped_memory_per_thread/30);
     //}
  }
}
/*===================================================================

  free gpu and mapped cpu memory

===================================================================*/
void GPUHelper::CudaFinalizeGPU(Options&options){
  if (num_gpus>0){
     #pragma omp parallel for schedule (static) num_threads(num_gpus)
     for (long int i=0; i<num_gpus; i++){
         long int thread = 0;
         #ifdef _OPENMP
           thread = omp_get_thread_num();
         #endif
         hipSetDevice(thread);
         Check_CUDA_Error(stdout,"hipSetDevice (free)");
         hipHostFree(tmp[thread]);
         Check_CUDA_Error(stdout,"cpu tmp (free)");
         hipFree(gpubuffer[thread]);
         Check_CUDA_Error(stdout,"gpu memory (free)");
     }
     free(tmp);
     free(gpubuffer);
     //for (long int i=0; i<num_cpus; i++){
     //    free(cpuarray[i]);
     //}
     //free(cpuarray);
  }
}

/**
 * dgemm assuming no tiling is necessary
 */
void GPUHelper::GPU_DGEMM(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){
  double*gpuA,*gpuB,*gpuC;
  hipMalloc((void**)&gpuA,m*k*sizeof(double));
  hipMalloc((void**)&gpuB,n*k*sizeof(double));
  hipMalloc((void**)&gpuC,m*n*sizeof(double));
  hipMemcpy(gpuA,A,m*k*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(gpuB,B,n*k*sizeof(double),hipMemcpyHostToDevice);
  hipblasDgemm(transa,transb,m,n,k,alpha,gpuA,lda,gpuB,ldb,beta,gpuC,ldc);
  hipMemcpy(C,gpuC,m*n*sizeof(double),hipMemcpyDeviceToHost);
  hipFree(gpuA);
  hipFree(gpuB);
  hipFree(gpuC);
}
/**
 * dgemm using a 2-dimensional tile - threaded versions for multiple gpus
 */
void GPUHelper::GPU_DGEMM_2DTile_nn_threaded_WithCpuStealing(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  throw PsiException("GPU_DGEMM_2DTile_nn_threaded_WithCpuStealing: not implemented",__FILE__,__LINE__);
//DPG commented out to remove statement unreachable warning
/*
  TilingWithCpuStealing((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);
  //Tiling((gpumemory-extraroom)/8L,max_mapped_memory/num_gpus/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  #pragma omp parallel num_threads(num_gpus+num_cpus)
  {

  long int thread = 0;
  #ifdef _OPENMP
    thread = omp_get_thread_num();
  #endif

  double*gpuA,*gpuB,*gpuC;
  // pointers to gpu memory
  if (thread<num_gpus){
     hipSetDevice(thread);
     gpuA = gpubuffer[thread];
     gpuB = gpubuffer[thread]+tilesizeM*tilesizeK;
     gpuC = gpubuffer[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;
  }
  // pointers to cpu memory
  else {
     gpuA = cpuarray[thread-num_gpus];
     gpuB = cpuarray[thread-num_gpus]+tilesizeMprime*tilesizeK;
     gpuC = cpuarray[thread-num_gpus]+tilesizeMprime*tilesizeK+tilesizeNprime*tilesizeK;
  }

  // cpu takes some of the 'N' tile
  if (StolenDimension=='N'){
     for (long int tm=0; tm<ntilesM; tm++){
         for (long int tk=0; tk<ntilesK; tk++){

             // this is for the gpus:
             if (thread<num_gpus){
                for (long int i=0; i<tilesizesK[tk]; i++){
                    C_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
                }
                hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
             
                for (long int tn=0; tn<ntilesN; tn++){
                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesN[tn]; i++){
                        C_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                    }
                    hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesM[tm]);
                    hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        C_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                    }
                }
             }
             // this if for any cpu cores that might be helping:
             else{
                for (long int i=0; i<tilesizesK[tk]; i++){
                    C_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,gpuA+i*tilesizesM[tm],1);
                }

                for (long int tn=0; tn<ntilesNprime; tn++){
                    if ((tm*ntilesNprime+tn)%num_cpus + num_gpus!=thread) continue;
                    for (long int i=0; i<tilesizesNprime[tn]; i++){
                        C_DCOPY(tilesizesK[tk],B+(NprimeOffSet+i+tn*tilesizeNprime)*ldb+tk*tilesizeK,1,gpuB+i*tilesizesK[tk],1);
                    }
                    F_DGEMM(transa,transb,tilesizesM[tm],tilesizesNprime[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesM[tm]);
                    for (long int j=0; j<tilesizesNprime[tn]; j++){
                        C_DAXPY(tilesizesM[tm],1.0,gpuC+j*tilesizesM[tm],1,C+(NprimeOffSet+j+tn*tilesizeNprime)*ldc+tm*tilesizeM,1);
                    }
                }
             }
         }
     }   
  }   
  // cpu takes some of the 'M' tile
  else if (StolenDimension=='M'){
     for (long int tn=0; tn<ntilesN; tn++){
         for (long int tk=0; tk<ntilesK; tk++){

             // this is for the gpus:
             if (thread<num_gpus){

                for (long int i=0; i<tilesizesN[tn]; i++){
                    C_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                }
                hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
             
                for (long int tm=0; tm<ntilesM; tm++){
                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesK[tk]; i++){
                        C_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
                    }
                    hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesM[tm]);
                    hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        C_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                    }
                }
             }
             // this if for any cpu cores that might be helping:
             else{
                for (long int i=0; i<tilesizesN[tn]; i++){
                    C_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,gpuB+i*tilesizesK[tk],1);
                }
             
                for (long int tm=0; tm<ntilesMprime; tm++){
                    if ((tm*ntilesN+tn)%num_cpus+num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesK[tk]; i++){
                        C_DCOPY(tilesizesMprime[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeMprime+MprimeOffSet,1,gpuA+i*tilesizesMprime[tm],1);
                    }
                    F_DGEMM(transa,transb,tilesizesMprime[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesMprime[tm],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesMprime[tm]);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        C_DAXPY(tilesizesMprime[tm],1.0,gpuC+j*tilesizesMprime[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeMprime+MprimeOffSet,1);
                    }
                }
             }
         }
     }   
  }
  else{
     if (thread<num_gpus){
        for (long int tm=0; tm<ntilesM; tm++){
            for (long int tk=0; tk<ntilesK; tk++){

                for (long int i=0; i<tilesizesK[tk]; i++){
                    C_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
                }
                hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                
                for (long int tn=0; tn<ntilesN; tn++){
                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesN[tn]; i++){
                        C_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                    }
                    hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesM[tm]);
                    hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        C_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                    }
                }
            }
        }
     }
  }

  }
  free(tilesizesMprime);
  free(tilesizesNprime);
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
*/
}

/**
 * dgemm using a 2-dimensional tile - threaded versions for multiple gpus
 */
void report_num_threads(int level)
{
    #pragma omp single
    {
        printf("Level %d: number of threads in the team - %d\n",
                  level, omp_get_num_threads());
    }
 }
void GPUHelper::GPU_DGEMM_2DTile_nn_threaded(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  Tiling((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  omp_set_nested(1);
  omp_set_dynamic(0);
  #pragma omp parallel for schedule (static) num_threads(num_gpus)
  for (long int mn=0; mn<ntilesM*ntilesN; mn++){
      long int thread = 0;
      #ifdef _OPENMP
        thread = omp_get_thread_num();
      #endif
      hipSetDevice(thread);

      // pointers to gpu memory
      double*gpuA = gpubuffer[thread];
      double*gpuB = gpubuffer[thread]+tilesizeM*tilesizeK*2;
      double*gpuC = gpubuffer[thread]+tilesizeM*tilesizeK*2+tilesizeN*tilesizeK*2;

      long int offsetA = tilesizeM * tilesizeK;
      long int offsetB = tilesizeN * tilesizeK;

      long int tn = mn%ntilesN;
      long int tm = (mn-tn)/ntilesN;
      hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));
      omp_set_nested(1);
      omp_set_dynamic(0);
if (interleaved_dgemm) {
      // create streams:
      hipStream_t stream1;
      hipStreamCreate(&stream1);
      hipEvent_t estart1,estop1;
      hipEventCreate(&estart1);
      hipEventCreate(&estop1);
      cublasSetKernelStream(stream1);

      hipStream_t stream2;
      hipStreamCreate(&stream2);
      hipEvent_t estart2,estop2;
      hipEventCreate(&estart2);
      hipEventCreate(&estop2);

      double start = omp_get_wtime();

      // need to transfer data for first tile
      for (long int i=0; i<tilesizesK[0]; i++){
          C_DCOPY(tilesizesM[tm],A+(i+0*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
      }
      hipMemcpyAsync(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[0]*sizeof(double),hipMemcpyHostToDevice,stream1);
      hipStreamSynchronize(stream1);
      for (long int i=0; i<tilesizesN[tn]; i++){
          C_DCOPY(tilesizesK[0],B+(i+tn*tilesizeN)*ldb+0*tilesizeK,1,tmp[thread]+i*tilesizesK[0],1);
      }
      hipMemcpyAsync(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[0]*sizeof(double),hipMemcpyHostToDevice,stream1);
      hipStreamSynchronize(stream1);

      for (long int tk=0; tk<ntilesK; tk++){

          #pragma omp parallel num_threads(2)
          {

              long int thread2 = omp_get_thread_num();
              if (thread2 == 0) {

                  double * A_curr = ( tk % 2 == 0 ) ? gpuA : gpuA + offsetA;
                  double * B_curr = ( tk % 2 == 0 ) ? gpuB : gpuB + offsetB;

                  hipEventRecord(estart1,stream1);
                      hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,A_curr,tilesizesM[tm],B_curr,tilesizesK[tk],1.0,gpuC,tilesizesM[tm]);
                      hipStreamSynchronize(stream1);
                  hipEventRecord(estop1,stream1);

              } else {
                  // only copy next tiles if we need them:
                  if ( tk < ntilesK - 1 ) {
                      double * A_next = ( tk % 2 == 0 ) ? gpuA + offsetA : gpuA;
                      double * B_next = ( tk % 2 == 0 ) ? gpuB + offsetB : gpuB;
                      hipEventRecord(estart2,stream2);
                          for (long int i=0; i<tilesizesK[tk+1]; i++){
                              C_DCOPY(tilesizesM[tm],A+(i+(tk+1)*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
                          }
                          hipMemcpyAsync(A_next,tmp[thread],tilesizesM[tm]*tilesizesK[tk+1]*sizeof(double),hipMemcpyHostToDevice,stream2);
                          hipStreamSynchronize(stream2);
                          for (long int i=0; i<tilesizesN[tn]; i++){
                              C_DCOPY(tilesizesK[tk+1],B+(i+tn*tilesizeN)*ldb+(tk+1)*tilesizeK,1,tmp[thread]+i*tilesizesK[tk+1],1);
                          }
                          hipMemcpyAsync(B_next,tmp[thread],tilesizesN[tn]*tilesizesK[tk+1]*sizeof(double),hipMemcpyHostToDevice,stream2);
                          hipStreamSynchronize(stream2);
                      hipEventRecord(estop2,stream2);
                  }
              }
          }
          hipDeviceSynchronize();
      }
      cublasSetKernelStream(NULL);
      hipEventDestroy(estart2);
      hipEventDestroy(estart1);
      hipEventDestroy(estop1);
      hipEventDestroy(estop2);
      hipStreamDestroy(stream1);
      hipStreamDestroy(stream2);
}else {
      // original version:
      for (long int tk=0; tk<ntilesK; tk++){
          for (long int i=0; i<tilesizesK[tk]; i++){
              C_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<tilesizesN[tn]; i++){
              C_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesK[tk],1.0,gpuC,tilesizesM[tm]);
      }
}
      omp_set_nested(0);
      omp_set_dynamic(1);
      hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
      for (long int j=0; j<tilesizesN[tn]; j++){
          C_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
      }
  }
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_nn(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc,int thread){

  TilingNoThread((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  hipSetDevice(thread);

  for (long int mn=0; mn<myntilesM[thread]*myntilesN[thread]; mn++){

      // pointers to gpu memory
      double*gpuA = gpubuffer[thread];
      double*gpuB = gpubuffer[thread]+mytilesizeM[thread]*mytilesizeK[thread];
      double*gpuC = gpubuffer[thread]+mytilesizeM[thread]*mytilesizeK[thread]+mytilesizeN[thread]*mytilesizeK[thread];

      long int tn = mn%myntilesN[thread];
      long int tm = (mn-tn)/myntilesN[thread];

      hipMemset((void*)gpuC,'\0',mytilesizesM[thread][tm]*mytilesizesN[thread][tn]*sizeof(double));
      for (long int tk=0; tk<myntilesK[thread]; tk++){

          for (long int i=0; i<mytilesizesK[thread][tk]; i++){
              C_DCOPY(mytilesizesM[thread][tm],A+(i+tk*mytilesizeK[thread])*lda+tm*mytilesizeM[thread],1,tmp[thread]+i*mytilesizesM[thread][tm],1);
          }
          hipMemcpy(gpuA,tmp[thread],mytilesizesM[thread][tm]*mytilesizesK[thread][tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<mytilesizesN[thread][tn]; i++){
              C_DCOPY(mytilesizesK[thread][tk],B+(i+tn*mytilesizeN[thread])*ldb+tk*mytilesizeK[thread],1,tmp[thread]+i*mytilesizesK[thread][tk],1);
          }
          hipMemcpy(gpuB,tmp[thread],mytilesizesN[thread][tn]*mytilesizesK[thread][tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,mytilesizesM[thread][tm],mytilesizesN[thread][tn],mytilesizesK[thread][tk],alpha,gpuA,mytilesizesM[thread][tm],gpuB,mytilesizesK[thread][tk],1.0,gpuC,mytilesizesM[thread][tm]);
      }
      hipMemcpy(tmp[thread],gpuC,mytilesizesN[thread][tn]*mytilesizesM[thread][tm]*sizeof(double),hipMemcpyDeviceToHost);
      for (long int j=0; j<mytilesizesN[thread][tn]; j++){
          C_DAXPY(mytilesizesM[thread][tm],1.0,tmp[thread]+j*mytilesizesM[thread][tm],1,C+(j+tn*mytilesizeN[thread])*ldc+tm*mytilesizeM[thread],1);
      }
  }
  free(mytilesizesM[thread]);
  free(mytilesizesN[thread]);
  free(mytilesizesK[thread]);
}
void GPUHelper::GPU_DGEMM_2DTile_nt_threaded_WithCpuStealing(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){


  throw PsiException("GPU_DGEMM_2DTile_nt_threaded_WithCpuStealing: not implemented",__FILE__,__LINE__);
//DPG commented out to remove statement unreachable warning
/*
  //Tiling((gpumemory-extraroom)/8L,max_mapped_memory/num_gpus/8L,m,n,k);
  TilingWithCpuStealing((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;


  #pragma omp parallel num_threads(num_gpus+num_cpus)
  {

  long int thread = 0;
  #ifdef _OPENMP
    thread = omp_get_thread_num();
  #endif

  double*gpuA,*gpuB,*gpuC;

  // pointers to gpu memory
  if (thread<num_gpus){
     hipSetDevice(thread);
     gpuA = gpubuffer[thread];
     gpuB = gpubuffer[thread]+tilesizeM*tilesizeK;
     gpuC = gpubuffer[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;
  }
  // pointers to cpu memory
  else {
     gpuA = cpuarray[thread-num_gpus];
     gpuB = cpuarray[thread-num_gpus]+tilesizeMprime*tilesizeK;
     gpuC = cpuarray[thread-num_gpus]+tilesizeMprime*tilesizeK+tilesizeNprime*tilesizeK;
  }

  // cpu takes some of the 'N' tile
  if (StolenDimension=='N'){
     for (long int tm=0; tm<ntilesM; tm++){
         for (long int tk=0; tk<ntilesK; tk++){
             if (thread<num_gpus){

                for (long int i=0; i<tilesizesK[tk]; i++){
                    C_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
                }
                hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);

                for (long int tn=0; tn<ntilesN; tn++){
                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesK[tk]; i++){
                        C_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
                    }
                    hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesN[tn],0.0,gpuC,tilesizesM[tm]);
                   hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                   for (long int j=0; j<tilesizesN[tn]; j++){
                       C_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                   }
                }

             }
             else{

                for (long int i=0; i<tilesizesK[tk]; i++){
                    C_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,gpuA+i*tilesizesM[tm],1);
                }

                for (long int tn=0; tn<ntilesNprime; tn++){
                    if ((tm*ntilesNprime+tn)%num_cpus+num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesK[tk]; i++){
                        C_DCOPY(tilesizesNprime[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeNprime+NprimeOffSet,1,gpuB+i*tilesizesNprime[tn],1);
                    }
                    F_DGEMM(transa,transb,tilesizesM[tm],tilesizesNprime[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesNprime[tn],0.0,gpuC,tilesizesM[tm]);
                    for (long int j=0; j<tilesizesNprime[tn]; j++){
                        C_DAXPY(tilesizesM[tm],1.0,gpuC+j*tilesizesM[tm],1,C+(j+tn*tilesizeNprime+NprimeOffSet)*ldc+tm*tilesizeM,1);
                    }
                }

             }
         }
     }
  }
  else if (StolenDimension=='M'){
     for (long int tn=0; tn<ntilesN; tn++){
         for (long int tk=0; tk<ntilesK; tk++){
             if (thread<num_gpus){

                for (long int i=0; i<tilesizesK[tk]; i++){
                    C_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
                }
                hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);

                for (long int tm=0; tm<ntilesM; tm++){
                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesK[tk]; i++){
                        C_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
                    }
                    hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);

                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesN[tn],0.0,gpuC,tilesizesM[tm]);
                    hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        C_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                    }
                }

             }
             else{

                for (long int i=0; i<tilesizesK[tk]; i++){
                    C_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,gpuB+i*tilesizesN[tn],1);
                }

                for (long int tm=0; tm<ntilesMprime; tm++){
                    if ((tm*ntilesN+tn)%num_cpus+num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesK[tk]; i++){
                        C_DCOPY(tilesizesMprime[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeMprime+MprimeOffSet,1,gpuA+i*tilesizesMprime[tm],1);
                    }

                    F_DGEMM(transa,transb,tilesizesMprime[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesMprime[tm],gpuB,tilesizesN[tn],0.0,gpuC,tilesizesMprime[tm]);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        C_DAXPY(tilesizesMprime[tm],1.0,gpuC+j*tilesizesMprime[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeMprime+MprimeOffSet,1);
                    }
                }

             }
         }
     }
  }
  else{
     for (long int tm=0; tm<ntilesM; tm++){
         for (long int tn=0; tn<ntilesN; tn++){
             if (thread<num_gpus){
                if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));
                for (long int tk=0; tk<ntilesK; tk++){
                    for (long int i=0; i<tilesizesK[tk]; i++){
                        C_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
                    }
                    hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    for (long int i=0; i<tilesizesK[tk]; i++){
                        C_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
                    }
                    hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesN[tn],1.0,gpuC,tilesizesM[tm]);
                }
                hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                for (long int j=0; j<tilesizesN[tn]; j++){
                    C_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                }

             }
         }
     }
  }
  }

  free(tilesizesNprime);
  free(tilesizesMprime);
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
*/
}
void GPUHelper::GPU_DGEMM_2DTile_nt_threaded(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  Tiling((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;


  omp_set_nested(1);
  omp_set_dynamic(0);
  #pragma omp parallel for schedule (static) num_threads(num_gpus)
  for (long int mn=0; mn<ntilesM*ntilesN; mn++){
      long int thread = 0;
      #ifdef _OPENMP
        thread = omp_get_thread_num();
      #endif
      hipSetDevice(thread);

      // pointers to gpu memory ... keep in mind that tilesizeK has been reduced by at least a factor of 2.
      double*gpuA = gpubuffer[thread];
      double*gpuB = gpubuffer[thread]+tilesizeM*tilesizeK*2;
      double*gpuC = gpubuffer[thread]+tilesizeM*tilesizeK*2+tilesizeN*tilesizeK*2;

      long int offsetA = tilesizeM * tilesizeK;
      long int offsetB = tilesizeN * tilesizeK;

      long int tn = mn%ntilesN;
      long int tm = (mn-tn)/ntilesN;
      hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));

      omp_set_nested(1);
      omp_set_dynamic(0);
if (interleaved_dgemm) {
      // create streams:
      hipStream_t stream1;
      hipStreamCreate(&stream1);
      hipEvent_t estart1,estop1;
      hipEventCreate(&estart1);
      hipEventCreate(&estop1);
      cublasSetKernelStream(stream1);

      hipStream_t stream2;
      hipStreamCreate(&stream2);
      hipEvent_t estart2,estop2;
      hipEventCreate(&estart2);
      hipEventCreate(&estop2);

      double start = omp_get_wtime();

      // need to transfer data for first tile
      for (long int i=0; i<tilesizesK[0]; i++){
          C_DCOPY(tilesizesM[tm],A+(i+0*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
      }
      hipMemcpyAsync(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[0]*sizeof(double),hipMemcpyHostToDevice,stream1);
      hipStreamSynchronize(stream1);
     for (long int i=0; i<tilesizesK[0]; i++){
          C_DCOPY(tilesizesN[tn],B+(i+0*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
      }
      hipMemcpyAsync(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[0]*sizeof(double),hipMemcpyHostToDevice,stream1);
      hipStreamSynchronize(stream1);
      for (long int tk=0; tk<ntilesK; tk++){
	#pragma omp parallel num_threads(2)
	{
              int thread2 = omp_get_thread_num();
              if (thread2 == 0) {

                  double * A_curr = ( tk % 2 == 0 ) ? gpuA : gpuA + offsetA;
                  double * B_curr = ( tk % 2 == 0 ) ? gpuB : gpuB + offsetB;

                  hipEventRecord(estart1,stream1);
                      hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,A_curr,tilesizesM[tm],B_curr,tilesizesN[tn],1.0,gpuC,tilesizesM[tm]);
                      hipStreamSynchronize(stream1);
                  hipEventRecord(estop1,stream1);

              } else {
                  // only copy next tiles if we need them:
                  if ( tk < ntilesK - 1) {
                      double * A_next = ( tk % 2 == 0 ) ? gpuA + offsetA : gpuA;
                      double * B_next = ( tk % 2 == 0 ) ? gpuB + offsetB : gpuB;
                      hipEventRecord(estart2,stream2);
                          for (long int i=0; i<tilesizesK[tk+1]; i++){
                              C_DCOPY(tilesizesM[tm],A+(i+(tk+1)*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
                          }
                          hipMemcpyAsync(A_next,tmp[thread],tilesizesM[tm]*tilesizesK[tk+1]*sizeof(double),hipMemcpyHostToDevice,stream2);
                          hipStreamSynchronize(stream2);
                          for (long int i=0; i<tilesizesK[(tk+1)]; i++){
                              C_DCOPY(tilesizesN[tn],B+(i+(tk+1)*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
                          }
                          hipMemcpyAsync(B_next,tmp[thread],tilesizesN[tn]*tilesizesK[tk+1]*sizeof(double),hipMemcpyHostToDevice,stream2);
                          hipStreamSynchronize(stream2);
                      hipEventRecord(estop2,stream2);
                  }
              }
          }
          hipDeviceSynchronize();
      }
      cublasSetKernelStream(NULL);
      hipEventDestroy(estart2);
      hipEventDestroy(estart1);
      hipEventDestroy(estop1);
      hipEventDestroy(estop2);
      hipStreamDestroy(stream1);
      hipStreamDestroy(stream2);
}else {
      // original version:
      for (long int tk=0; tk<ntilesK; tk++){
          for (long int i=0; i<tilesizesK[tk]; i++){
              C_DCOPY(tilesizesM[tm],A+(i+tk*tilesizeK)*lda+tm*tilesizeM,1,tmp[thread]+i*tilesizesM[tm],1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<tilesizesK[tk]; i++){
              C_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesM[tm],gpuB,tilesizesN[tn],1.0,gpuC,tilesizesM[tm]);
      }
}
      omp_set_nested(0);
      omp_set_dynamic(1);
      hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
      for (long int j=0; j<tilesizesN[tn]; j++){
          C_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
      }
  }
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_nt(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc,int thread){

  TilingNoThread((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  hipSetDevice(thread);

  for (long int mn=0; mn<myntilesM[thread]*myntilesN[thread]; mn++){

      // pointers to gpu memory
      double*gpuA = gpubuffer[thread];
      double*gpuB = gpubuffer[thread]+mytilesizeM[thread]*mytilesizeK[thread];
      double*gpuC = gpubuffer[thread]+mytilesizeM[thread]*mytilesizeK[thread]+mytilesizeN[thread]*mytilesizeK[thread];

      long int tn = mn%myntilesN[thread];
      long int tm = (mn-tn)/myntilesN[thread];

      hipMemset((void*)gpuC,'\0',mytilesizesM[thread][tm]*mytilesizesN[thread][tn]*sizeof(double));
      for (long int tk=0; tk<myntilesK[thread]; tk++){
          for (long int i=0; i<mytilesizesK[thread][tk]; i++){
              C_DCOPY(mytilesizesM[thread][tm],A+(i+tk*mytilesizeK[thread])*lda+tm*mytilesizeM[thread],1,tmp[thread]+i*mytilesizesM[thread][tm],1);
          }
          hipMemcpy(gpuA,tmp[thread],mytilesizesM[thread][tm]*mytilesizesK[thread][tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<mytilesizesK[thread][tk]; i++){
              C_DCOPY(mytilesizesN[thread][tn],B+(i+tk*mytilesizeK[thread])*ldb+tn*mytilesizeN[thread],1,tmp[thread]+i*mytilesizesN[thread][tn],1);
          }
          hipMemcpy(gpuB,tmp[thread],mytilesizesN[thread][tn]*mytilesizesK[thread][tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,mytilesizesM[thread][tm],mytilesizesN[thread][tn],mytilesizesK[thread][tk],alpha,gpuA,mytilesizesM[thread][tm],gpuB,mytilesizesN[thread][tn],1.0,gpuC,mytilesizesM[thread][tm]);
      }
      hipMemcpy(tmp[thread],gpuC,mytilesizesN[thread][tn]*mytilesizesM[thread][tm]*sizeof(double),hipMemcpyDeviceToHost);
      for (long int j=0; j<mytilesizesN[thread][tn]; j++){
          C_DAXPY(mytilesizesM[thread][tm],1.0,tmp[thread]+j*mytilesizesM[thread][tm],1,C+(j+tn*mytilesizeN[thread])*ldc+tm*mytilesizeM[thread],1);
      }
  }
  free(mytilesizesM[thread]);
  free(mytilesizesN[thread]);
  free(mytilesizesK[thread]);
}
void GPUHelper::GPU_DGEMM_2DTile_tn_threaded_WithCpuStealing(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  throw PsiException("GPU_DGEMM_2DTile_tn_threaded_WithCpuStealing: not implemented",__FILE__,__LINE__);
//DPG commented out to remove statement unreachable warning
/*
  //Tiling((gpumemory-extraroom)/8L,max_mapped_memory/num_gpus/8L,m,n,k);
  TilingWithCpuStealing((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  #pragma omp parallel num_threads(num_gpus+num_cpus)
  {

  long int thread = 0;
  #ifdef _OPENMP
    thread = omp_get_thread_num();
  #endif

  double*gpuA,*gpuB,*gpuC;

  // pointers to gpu memory
  if (thread<num_gpus){
     hipSetDevice(thread);
     gpuA = gpubuffer[thread];
     gpuB = gpubuffer[thread]+tilesizeM*tilesizeK;
     gpuC = gpubuffer[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;
  }
  // pointers to cpu memory
  else {
     gpuA = cpuarray[thread-num_gpus];
     gpuB = cpuarray[thread-num_gpus]+tilesizeMprime*tilesizeK;
     gpuC = cpuarray[thread-num_gpus]+tilesizeMprime*tilesizeK+tilesizeNprime*tilesizeK;
  }

  // cpu takes some of the 'N' tile
  StolenDimension=' ';
  if (StolenDimension=='N'){
     for (long int tm=0; tm<ntilesM; tm++){
         for (long int tk=0; tk<ntilesK; tk++){
             if (thread<num_gpus){
                for (long int i=0; i<tilesizesM[tm]; i++){
                    C_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                }
                hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);

                for (long int tn=0; tn<ntilesN; tn++){

                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesN[tn]; i++){
                        C_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                    }
                    hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesM[tm]);
                    hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        C_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                    }
                }
             }
             else{
                for (long int i=0; i<tilesizesM[tm]; i++){
                    C_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,gpuA+i*tilesizesK[tk],1);
                }

                for (long int tn=0; tn<ntilesNprime; tn++){

                    if ((tm*ntilesNprime+tn)%num_cpus+num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesNprime[tn]; i++){
                        C_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeNprime+NprimeOffSet)*ldb+tk*tilesizeK,1,gpuB+i*tilesizesK[tk],1);
                    }
                    F_DGEMM(transa,transb,tilesizesM[tm],tilesizesNprime[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesM[tm]);
                    for (long int j=0; j<tilesizesNprime[tn]; j++){
                        C_DAXPY(tilesizesM[tm],1.0,gpuC+j*tilesizesM[tm],1,C+(j+tn*tilesizeNprime+NprimeOffSet)*ldc+tm*tilesizeM,1);
                    }
                }
             }
         }
     }
  }
  else if (StolenDimension=='M'){
     for (long int tn=0; tn<ntilesN; tn++){
         for (long int tk=0; tk<ntilesK; tk++){
             if (thread<num_gpus){
                for (long int i=0; i<tilesizesN[tn]; i++){
                    C_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                }
                hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);

                for (long int tm=0; tm<ntilesM; tm++){

                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesM[tm]; i++){
                        C_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                    }
                    hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesM[tm]);
                    hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        C_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                    }
                }
             }
             else{
                for (long int i=0; i<tilesizesN[tn]; i++){
                    C_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,gpuB+i*tilesizesK[tk],1);
                }

                for (long int tm=0; tm<ntilesMprime; tm++){

                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesMprime[tm]; i++){
                        C_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeMprime+MprimeOffSet)*lda+tk*tilesizeK,1,gpuA+i*tilesizesK[tk],1);
                    }
                    F_DGEMM(transa,transb,tilesizesMprime[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesK[tk],0.0,gpuC,tilesizesMprime[tm]);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        C_DAXPY(tilesizesMprime[tm],1.0,gpuC+j*tilesizesMprime[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeMprime+MprimeOffSet,1);
                    }
                }
             }
         }
     }
  }
  else{
     if (thread<num_gpus){
        for (long int tm=0; tm<ntilesM; tm++){
            for (long int tn=0; tn<ntilesN; tn++){
                hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));
                for (long int tk=0; tk<ntilesK; tk++){
                    for (long int i=0; i<tilesizesM[tm]; i++){
                        C_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                    }
                    hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    for (long int i=0; i<tilesizesN[tn]; i++){
                        C_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                    }
                    hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesK[tk],1.0,gpuC,tilesizesM[tm]);
                }
                hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                for (long int j=0; j<tilesizesN[tn]; j++){
                    C_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                }

            }
        }
     }
  }


  }
  free(tilesizesMprime);
  free(tilesizesNprime);
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
*/
}
void GPUHelper::GPU_DGEMM_2DTile_tn_threaded(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  Tiling((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  #pragma omp parallel for schedule (static) num_threads(num_gpus)
  for (long int mn=0; mn<ntilesM*ntilesN; mn++){
      long int thread = 0;
      #ifdef _OPENMP
        thread = omp_get_thread_num();
      #endif
      hipSetDevice(thread);

      // pointers to gpu memory
      double*gpuA = gpubuffer[thread];
      double*gpuB = gpubuffer[thread]+tilesizeM*tilesizeK*2;
      double*gpuC = gpubuffer[thread]+tilesizeM*tilesizeK*2+tilesizeN*tilesizeK*2;

      long int offsetA = tilesizeM * tilesizeK;
      long int offsetB = tilesizeN * tilesizeK;

      long int tn = mn%ntilesN;
      long int tm = (mn-tn)/ntilesN;

      hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));
    
      omp_set_nested(1);
      omp_set_dynamic(0);
if (interleaved_dgemm) {
      // create streams:
      hipStream_t stream1;
      hipStreamCreate(&stream1);
      hipEvent_t estart1,estop1;
      hipEventCreate(&estart1);
      hipEventCreate(&estop1);
      cublasSetKernelStream(stream1);

      hipStream_t stream2;
      hipStreamCreate(&stream2);
      hipEvent_t estart2,estop2;
      hipEventCreate(&estart2);
      hipEventCreate(&estop2);

      double start = omp_get_wtime();

      // need to transfer data for first tile
      for (long int i=0; i<tilesizesM[tm]; i++){
          C_DCOPY(tilesizesK[0],A+(i+tm*tilesizeM)*lda+0*tilesizeK,1,tmp[thread]+i*tilesizesK[0],1);
      }
      hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[0]*sizeof(double),hipMemcpyHostToDevice);
      hipStreamSynchronize(stream1);
      for (long int i=0; i<tilesizesN[tn]; i++){
          C_DCOPY(tilesizesK[0],B+(i+tn*tilesizeN)*ldb+0*tilesizeK,1,tmp[thread]+i*tilesizesK[0],1);
      }
      hipMemcpyAsync(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[0]*sizeof(double),hipMemcpyHostToDevice,stream1);
      hipStreamSynchronize(stream1);

      for (long int tk=0; tk<ntilesK; tk++){

          #pragma omp parallel num_threads(2)
          {

              long int thread2 = omp_get_thread_num();
              if (thread2 == 0) {

                  double * A_curr = ( tk % 2 == 0 ) ? gpuA : gpuA + offsetA;
                  double * B_curr = ( tk % 2 == 0 ) ? gpuB : gpuB + offsetB;

                  hipEventRecord(estart1,stream1);
                      hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,A_curr,tilesizesK[tk],B_curr,tilesizesK[tk],1.0,gpuC,tilesizesM[tm]);
                      hipStreamSynchronize(stream1);
                  hipEventRecord(estop1,stream1);

              } else {
                  // only copy next tiles if we need them:
                  if ( tk < ntilesK - 1 ) {
                      double * A_next = ( tk % 2 == 0 ) ? gpuA + offsetA : gpuA;
                      double * B_next = ( tk % 2 == 0 ) ? gpuB + offsetB : gpuB;
                      hipEventRecord(estart2,stream2);
                          for (long int i=0; i<tilesizesM[tm]; i++){
                              C_DCOPY(tilesizesK[tk+1],A+(i+tm*tilesizeM)*lda+(tk+1)*tilesizeK,1,tmp[thread]+i*tilesizesK[tk+1],1);
                          }
                          hipMemcpyAsync(A_next,tmp[thread],tilesizesM[tm]*tilesizesK[tk+1]*sizeof(double),hipMemcpyHostToDevice,stream2);
                          hipStreamSynchronize(stream2);
                          for (long int i=0; i<tilesizesN[tn]; i++){
                              C_DCOPY(tilesizesK[tk+1],B+(i+tn*tilesizeN)*ldb+(tk+1)*tilesizeK,1,tmp[thread]+i*tilesizesK[tk+1],1);
                          }
                          hipMemcpyAsync(B_next,tmp[thread],tilesizesN[tn]*tilesizesK[tk+1]*sizeof(double),hipMemcpyHostToDevice,stream2);
                          hipStreamSynchronize(stream2);
                      hipEventRecord(estop2,stream2);
                      //while( hipEventQuery(estop) == hipErrorNotReady );


                  }
              }
              hipDeviceSynchronize();
// TODO: something is wrong with this one ... how to fix ... 
          }
      }
      cublasSetKernelStream(NULL);
      hipEventDestroy(estart2);
      hipEventDestroy(estart1);
      hipEventDestroy(estop1);
      hipEventDestroy(estop2);
      hipStreamDestroy(stream1);
      hipStreamDestroy(stream2);
}else {
      // original version:
      for (long int tk=0; tk<ntilesK; tk++){
          for (long int i=0; i<tilesizesM[tm]; i++){
              C_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<tilesizesN[tn]; i++){
              C_DCOPY(tilesizesK[tk],B+(i+tn*tilesizeN)*ldb+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesK[tk],1.0,gpuC,tilesizesM[tm]);
      }
}
      omp_set_nested(0);
      omp_set_dynamic(1);
      hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
      for (long int j=0; j<tilesizesN[tn]; j++){
          C_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
      }
  }
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_tn(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc,int thread){

  TilingNoThread((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  hipSetDevice(thread);

  for (long int mn=0; mn<myntilesM[thread]*myntilesN[thread]; mn++){

      // pointers to gpu memory
      double*gpuA = gpubuffer[thread];
      double*gpuB = gpubuffer[thread]+mytilesizeM[thread]*mytilesizeK[thread];
      double*gpuC = gpubuffer[thread]+mytilesizeM[thread]*mytilesizeK[thread]+mytilesizeN[thread]*mytilesizeK[thread];

      long int tn = mn%myntilesN[thread];
      long int tm = (mn-tn)/myntilesN[thread];

      hipMemset((void*)gpuC,'\0',mytilesizesM[thread][tm]*mytilesizesN[thread][tn]*sizeof(double));
      for (long int tk=0; tk<myntilesK[thread]; tk++){
          for (long int i=0; i<mytilesizesM[thread][tm]; i++){
              C_DCOPY(mytilesizesK[thread][tk],A+(i+tm*mytilesizeM[thread])*lda+tk*mytilesizeK[thread],1,tmp[thread]+i*mytilesizesK[thread][tk],1);
          }
          hipMemcpy(gpuA,tmp[thread],mytilesizesM[thread][tm]*mytilesizesK[thread][tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<mytilesizesN[thread][tn]; i++){
              C_DCOPY(mytilesizesK[thread][tk],B+(i+tn*mytilesizeN[thread])*ldb+tk*mytilesizeK[thread],1,tmp[thread]+i*mytilesizesK[thread][tk],1);
          }
          hipMemcpy(gpuB,tmp[thread],mytilesizesN[thread][tn]*mytilesizesK[thread][tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,mytilesizesM[thread][tm],mytilesizesN[thread][tn],mytilesizesK[thread][tk],alpha,gpuA,mytilesizesK[thread][tk],gpuB,mytilesizesK[thread][tk],1.0,gpuC,mytilesizesM[thread][tm]);
      }
      hipMemcpy(tmp[thread],gpuC,mytilesizesN[thread][tn]*mytilesizesM[thread][tm]*sizeof(double),hipMemcpyDeviceToHost);
      for (long int j=0; j<mytilesizesN[thread][tn]; j++){
          C_DAXPY(mytilesizesM[thread][tm],1.0,tmp[thread]+j*mytilesizesM[thread][tm],1,C+(j+tn*mytilesizeN[thread])*ldc+tm*mytilesizeM[thread],1);
      }
  }
  free(mytilesizesM[thread]);
  free(mytilesizesN[thread]);
  free(mytilesizesK[thread]);
}
void GPUHelper::GPU_DGEMM_2DTile_tt_threaded_WithCpuStealing(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  throw PsiException("GPU_DGEMM_2DTile_tt_threaded_WithCpuStealing: not implemented",__FILE__,__LINE__);
//DPG commented out to remove statement unreachable warning
/*
  //Tiling((gpumemory-extraroom)/8L,max_mapped_memory/num_gpus/8L,m,n,k);
  TilingWithCpuStealing((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;


  #pragma omp parallel num_threads(num_gpus+num_cpus)
  {

  long int thread = 0;
  #ifdef _OPENMP
    thread = omp_get_thread_num();
  #endif

  double*gpuA,*gpuB,*gpuC;

  // pointers to gpu memory
  if (thread<num_gpus){
     hipSetDevice(thread);
     gpuA = gpubuffer[thread];
     gpuB = gpubuffer[thread]+tilesizeM*tilesizeK;
     gpuC = gpubuffer[thread]+tilesizeM*tilesizeK+tilesizeN*tilesizeK;
  }
  // pointers to cpu memory
  else {
     gpuA = cpuarray[thread-num_gpus];
     gpuB = cpuarray[thread-num_gpus]+tilesizeMprime*tilesizeK;
     gpuC = cpuarray[thread-num_gpus]+tilesizeMprime*tilesizeK+tilesizeNprime*tilesizeK;
  }

  // cpu takes some of the 'N' tile
  StolenDimension=' ';
  if (StolenDimension=='N'){
     for (long int tm=0; tm<ntilesM; tm++){
         for (long int tk=0; tk<ntilesK; tk++){
             if (thread<num_gpus){
                for (long int i=0; i<tilesizesM[tm]; i++){
                    C_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                }
                hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                for (long int tn=0; tn<ntilesN; tn++){
                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesK[tk]; i++){
                        C_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
                    }
                    hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesN[tn],0.0,gpuC,tilesizesM[tm]);
                    hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        C_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                    }
                }
             }
             else{
                for (long int i=0; i<tilesizesM[tm]; i++){
                    C_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,gpuA+i*tilesizesK[tk],1);
                }
                for (long int tn=0; tn<ntilesNprime; tn++){
                    if ((tm*ntilesNprime+tn)%num_cpus+num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesK[tk]; i++){
                        C_DCOPY(tilesizesNprime[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeNprime+NprimeOffSet,1,gpuB+i*tilesizesN[tn],1);
                    }
                    F_DGEMM(transa,transb,tilesizesM[tm],tilesizesNprime[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesNprime[tn],0.0,gpuC,tilesizesM[tm]);
                    for (long int j=0; j<tilesizesNprime[tn]; j++){
                        C_DAXPY(tilesizesM[tm],1.0,gpuC+j*tilesizesM[tm],1,C+(j+tn*tilesizeNprime+NprimeOffSet)*ldc+tm*tilesizeM,1);
                    }
                }
             }
         }
     }
  }
  else if (StolenDimension=='M'){
     for (long int tn=0; tn<ntilesN; tn++){
         for (long int tk=0; tk<ntilesK; tk++){
             if (thread<num_gpus){
                for (long int i=0; i<tilesizesK[tk]; i++){
                    C_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
                }
                hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                for (long int tm=0; tm<ntilesM; tm++){

                    if ((tm*ntilesN+tn)%num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesM[tm]; i++){
                        C_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                    }
                    hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesN[tn],0.0,gpuC,tilesizesM[tm]);
                    hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        C_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                    }
                }
             }
             else{
                for (long int i=0; i<tilesizesK[tk]; i++){
                    C_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,gpuB+i*tilesizesN[tn],1);
                }
                for (long int tm=0; tm<ntilesMprime; tm++){

                    if ((tm*ntilesN+tn)%num_cpus+num_gpus!=thread) continue;

                    for (long int i=0; i<tilesizesMprime[tm]; i++){
                        C_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeMprime+MprimeOffSet)*lda+tk*tilesizeK,1,gpuA+i*tilesizesK[tk],1);
                    }
                    F_DGEMM(transa,transb,tilesizesMprime[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesN[tn],0.0,gpuC,tilesizesMprime[tm]);
                    for (long int j=0; j<tilesizesN[tn]; j++){
                        C_DAXPY(tilesizesMprime[tm],1.0,gpuC+j*tilesizesMprime[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeMprime+MprimeOffSet,1);
                    }
                }
             }
         }
     }
  }
  else{
     if (thread<num_gpus){
        for (long int tm=0; tm<ntilesM; tm++){
            for (long int tn=0; tn<ntilesN; tn++){
                hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));
                for (long int tk=0; tk<ntilesK; tk++){
                    for (long int i=0; i<tilesizesM[tm]; i++){
                        C_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
                    }
                    hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    for (long int i=0; i<tilesizesK[tk]; i++){
                        C_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
                    }
                    hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
                    hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesN[tn],1.0,gpuC,tilesizesM[tm]);
                }
                hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
                for (long int j=0; j<tilesizesN[tn]; j++){
                    C_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
                }
            }
        }
     }
  }

  }

  free(tilesizesMprime);
  free(tilesizesNprime);
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
*/
}
// TODO: not thoroughly tested yet.
void GPUHelper::GPU_DGEMM_2DTile_tt_threaded(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc){

  Tiling((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  #pragma omp parallel for schedule (static) num_threads(num_gpus)
  for (long int mn=0; mn<ntilesM*ntilesN; mn++){
      long int thread = 0;
      #ifdef _OPENMP
        thread = omp_get_thread_num();
      #endif
      hipSetDevice(thread);

      // pointers to gpu memory
      double*gpuA = gpubuffer[thread];
      double*gpuB = gpubuffer[thread]+tilesizeM*tilesizeK*2;
      double*gpuC = gpubuffer[thread]+tilesizeM*tilesizeK*2+tilesizeN*tilesizeK*2;

      long int offsetA = tilesizeM * tilesizeK;
      long int offsetB = tilesizeN * tilesizeK;

      long int tn = mn%ntilesN;
      long int tm = (mn-tn)/ntilesN;

      hipMemset((void*)gpuC,'\0',tilesizesM[tm]*tilesizesN[tn]*sizeof(double));

      // create streams:
      omp_set_nested(1);
      omp_set_dynamic(0);
if (interleaved_dgemm) {
      // create streams:
      hipStream_t stream1;
      hipStreamCreate(&stream1);
      hipEvent_t estart1,estop1;
      hipEventCreate(&estart1);
      hipEventCreate(&estop1);
      cublasSetKernelStream(stream1);

      hipStream_t stream2;
      hipStreamCreate(&stream2);
      hipEvent_t estart2,estop2;
      hipEventCreate(&estart2);
      hipEventCreate(&estop2);

      double start = omp_get_wtime();

      // need to transfer data for first tile
      for (long int i=0; i<tilesizesM[tm]; i++){
          C_DCOPY(tilesizesK[0],A+(i+tm*tilesizeM)*lda+0*tilesizeK,1,tmp[thread]+i*tilesizesK[0],1);
      }
      hipMemcpyAsync(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[0]*sizeof(double),hipMemcpyHostToDevice,stream1);
      hipStreamSynchronize(stream1);
      for (long int i=0; i<tilesizesK[0]; i++){
          C_DCOPY(tilesizesN[tn],B+(i+0*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
      }
      hipMemcpyAsync(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[0]*sizeof(double),hipMemcpyHostToDevice,stream1);
      hipStreamSynchronize(stream1);

      for (long int tk=0; tk<ntilesK; tk++){

          #pragma omp parallel num_threads(2)
          {

              long int thread2 = omp_get_thread_num();
              if (thread2 == 0) {

                  double * A_curr = ( tk % 2 == 0 ) ? gpuA : gpuA + offsetA;
                  double * B_curr = ( tk % 2 == 0 ) ? gpuB : gpuB + offsetB;

                  hipEventRecord(estart1,stream1);
                      hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,A_curr,tilesizesK[tk],B_curr,tilesizesN[tn],1.0,gpuC,tilesizesM[tm]);
                      hipStreamSynchronize(stream1);
                  hipEventRecord(estop1,stream1);

              } else {
                  // only copy next tiles if we need them:
                  if ( tk < ntilesK - 1 ) {
                      double * A_next = ( tk % 2 == 0 ) ? gpuA + offsetA : gpuA;
                      double * B_next = ( tk % 2 == 0 ) ? gpuB + offsetB : gpuB;
                      hipEventRecord(estart2,stream2);
                          for (long int i=0; i<tilesizesM[tm]; i++){
                              C_DCOPY(tilesizesK[tk+1],A+(i+tm*tilesizeM)*lda+(tk+1)*tilesizeK,1,tmp[thread]+i*tilesizesK[tk+1],1);
                          }
                          hipMemcpyAsync(A_next,tmp[thread],tilesizesM[tm]*tilesizesK[tk+1]*sizeof(double),hipMemcpyHostToDevice,stream2);
                          hipStreamSynchronize(stream2);
                          for (long int i=0; i<tilesizesK[tk+1]; i++){
                              C_DCOPY(tilesizesN[tn],B+(i+(tk+1)*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
                          }
                          hipMemcpyAsync(B_next,tmp[thread],tilesizesN[tn]*tilesizesK[tk+1]*sizeof(double),hipMemcpyHostToDevice,stream2);
                          hipStreamSynchronize(stream2);
                      hipEventRecord(estop2,stream2);
                  }
              }
          }
          hipDeviceSynchronize();
      }
      cublasSetKernelStream(NULL);
      hipEventDestroy(estart2);
      hipEventDestroy(estart1);
      hipEventDestroy(estop1);
      hipEventDestroy(estop2);
      hipStreamDestroy(stream1);
      hipStreamDestroy(stream2);
}else {
      // original version:
      for (long int tk=0; tk<ntilesK; tk++){
          for (long int i=0; i<tilesizesM[tm]; i++){
              C_DCOPY(tilesizesK[tk],A+(i+tm*tilesizeM)*lda+tk*tilesizeK,1,tmp[thread]+i*tilesizesK[tk],1);
          }
          hipMemcpy(gpuA,tmp[thread],tilesizesM[tm]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<tilesizesK[tk]; i++){
              C_DCOPY(tilesizesN[tn],B+(i+tk*tilesizeK)*ldb+tn*tilesizeN,1,tmp[thread]+i*tilesizesN[tn],1);
          }
          hipMemcpy(gpuB,tmp[thread],tilesizesN[tn]*tilesizesK[tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,tilesizesM[tm],tilesizesN[tn],tilesizesK[tk],alpha,gpuA,tilesizesK[tk],gpuB,tilesizesN[tn],1.0,gpuC,tilesizesM[tm]);
      }
}
      omp_set_nested(0);
      omp_set_dynamic(1);
      hipMemcpy(tmp[thread],gpuC,tilesizesN[tn]*tilesizesM[tm]*sizeof(double),hipMemcpyDeviceToHost);
      for (long int j=0; j<tilesizesN[tn]; j++){
          C_DAXPY(tilesizesM[tm],1.0,tmp[thread]+j*tilesizesM[tm],1,C+(j+tn*tilesizeN)*ldc+tm*tilesizeM,1);
      }
  }
  free(tilesizesM);
  free(tilesizesN);
  free(tilesizesK);
}
void GPUHelper::GPU_DGEMM_2DTile_tt(char transa,char transb,long int m,long int n,long int k,double alpha,double*A,long int lda,double*B,long int ldb,double beta,double*C,long int ldc,int thread){

  TilingNoThread((gpumemory-extraroom)/8L,max_mapped_memory_per_thread/8L,m,n,k);

  // initialize result
  if (beta==0.0) 
     memset((void*)C,'\0',n*ldc*sizeof(double));
  else           
     for (long int i=0; i<n*ldc; i++) C[i] *= beta;

  hipSetDevice(thread);

  for (long int mn=0; mn<myntilesM[thread]*myntilesN[thread]; mn++){

      // pointers to gpu memory
      double*gpuA = gpubuffer[thread];
      double*gpuB = gpubuffer[thread]+mytilesizeM[thread]*mytilesizeK[thread];
      double*gpuC = gpubuffer[thread]+mytilesizeM[thread]*mytilesizeK[thread]+mytilesizeN[thread]*mytilesizeK[thread];

      long int tn = mn%myntilesN[thread];
      long int tm = (mn-tn)/myntilesN[thread];

      hipMemset((void*)gpuC,'\0',mytilesizesM[thread][tm]*mytilesizesN[thread][tn]*sizeof(double));
      for (long int tk=0; tk<myntilesK[thread]; tk++){
          for (long int i=0; i<mytilesizesM[thread][tm]; i++){
              C_DCOPY(mytilesizesK[thread][tk],A+(i+tm*mytilesizeM[thread])*lda+tk*mytilesizeK[thread],1,tmp[thread]+i*mytilesizesK[thread][tk],1);
          }
          hipMemcpy(gpuA,tmp[thread],mytilesizesM[thread][tm]*mytilesizesK[thread][tk]*sizeof(double),hipMemcpyHostToDevice);
          for (long int i=0; i<mytilesizesK[thread][tk]; i++){
              C_DCOPY(mytilesizesN[thread][tn],B+(i+tk*mytilesizeK[thread])*ldb+tn*mytilesizeN[thread],1,tmp[thread]+i*mytilesizesN[thread][tn],1);
          }
          hipMemcpy(gpuB,tmp[thread],mytilesizesN[thread][tn]*mytilesizesK[thread][tk]*sizeof(double),hipMemcpyHostToDevice);
          hipblasDgemm(transa,transb,mytilesizesM[thread][tm],mytilesizesN[thread][tn],mytilesizesK[thread][tk],alpha,gpuA,mytilesizesK[thread][tk],gpuB,mytilesizesN[thread][tn],1.0,gpuC,mytilesizesM[thread][tm]);
      }
      hipMemcpy(tmp[thread],gpuC,mytilesizesN[thread][tn]*mytilesizesM[thread][tm]*sizeof(double),hipMemcpyDeviceToHost);
      for (long int j=0; j<mytilesizesN[thread][tn]; j++){
          C_DAXPY(mytilesizesM[thread][tm],1.0,tmp[thread]+j*mytilesizesM[thread][tm],1,C+(j+tn*mytilesizeN[thread])*ldc+tm*mytilesizeM[thread],1);
      }
  }
  free(mytilesizesM[thread]);
  free(mytilesizesN[thread]);
  free(mytilesizesK[thread]);
}

void GPUHelper::TilingNoThread(long int mem1,long int mem2,long int m,long int n,long int k){

  long int thread = 0;
  #ifdef _OPENMP
    thread = omp_get_thread_num();
  #endif

  // first tile according to how much space is on gpu
  mytilesizeN[thread] = n;
  mytilesizeM[thread] = m;
  mytilesizeK[thread] = k;
  myntilesM[thread]=myntilesN[thread]=myntilesK[thread]=1L;
  while(mytilesizeN[thread]*mytilesizeM[thread]+mytilesizeK[thread]*(mytilesizeN[thread]+mytilesizeM[thread])>mem1){
     if (mytilesizeN[thread]>mytilesizeM[thread]){
        if (mytilesizeN[thread]>mytilesizeK[thread]){
           myntilesN[thread]++;
           mytilesizeN[thread] = n/myntilesN[thread];
           if (n/myntilesN[thread]<(double)n/myntilesN[thread]) mytilesizeN[thread]++;
        }
        else{
           myntilesK[thread]++;
           mytilesizeK[thread] = k/myntilesK[thread];
           if (k/myntilesK[thread]<(double)k/myntilesK[thread]) mytilesizeK[thread]++;
        }
     }
     else{
        if (mytilesizeM[thread]>mytilesizeK[thread]){
           myntilesM[thread]++;
           mytilesizeM[thread] = m/myntilesM[thread];
           if (m/myntilesM[thread]<(double)m/myntilesM[thread]) mytilesizeM[thread]++;
        }
        else{
           myntilesK[thread]++;
           mytilesizeK[thread] = k/myntilesK[thread];
           if (k/myntilesK[thread]<(double)k/myntilesK[thread]) mytilesizeK[thread]++;
        }
     }
  }

  // ensure each block of A, B, and C will fit in the temporary CPU buffer
  while(mytilesizeN[thread]*mytilesizeM[thread]>mem2){
     if (mytilesizeN[thread]>mytilesizeM[thread]){
        myntilesN[thread]++;
        mytilesizeN[thread] = n/myntilesN[thread];
        if (n/myntilesN[thread]<(double)n/myntilesN[thread]) mytilesizeN[thread]++;
     }
     else{
        myntilesM[thread]++;
        mytilesizeM[thread] = m/myntilesM[thread];
        if (m/myntilesM[thread]<(double)m/myntilesM[thread]) mytilesizeM[thread]++;
     }
  }

  while(mytilesizeN[thread]*mytilesizeK[thread]>mem2){
     if (mytilesizeN[thread]>mytilesizeK[thread]){
        myntilesN[thread]++;
        mytilesizeN[thread] = n/myntilesN[thread];
        if (n/myntilesN[thread]<(double)n/myntilesN[thread]) mytilesizeN[thread]++;
     }
     else{
        myntilesK[thread]++;
        mytilesizeK[thread] = k/myntilesK[thread];
        if (k/myntilesK[thread]<(double)k/myntilesK[thread]) mytilesizeK[thread]++;
     }
  }
  while(mytilesizeK[thread]*mytilesizeM[thread]>mem2){
     if (mytilesizeK[thread]>mytilesizeM[thread]){
        myntilesK[thread]++;
        mytilesizeK[thread] = k/myntilesK[thread];
        if (k/myntilesK[thread]<(double)k/myntilesK[thread]) mytilesizeK[thread]++;
     }
     else{
        myntilesM[thread]++;
        mytilesizeM[thread] = m/myntilesM[thread];
        if (m/myntilesM[thread]<(double)m/myntilesM[thread]) mytilesizeM[thread]++;
     }
  }

  mylasttileN[thread] = n - (myntilesN[thread]-1L)*mytilesizeN[thread];
  mylasttileM[thread] = m - (myntilesM[thread]-1L)*mytilesizeM[thread];
  mylasttileK[thread] = k - (myntilesK[thread]-1L)*mytilesizeK[thread];

  mytilesizesM[thread] = (long int*)malloc(myntilesM[thread]*sizeof(long int));
  mytilesizesN[thread] = (long int*)malloc(myntilesN[thread]*sizeof(long int));
  mytilesizesK[thread] = (long int*)malloc(myntilesK[thread]*sizeof(long int));
  for (long int i=0; i<myntilesM[thread]-1L; i++) mytilesizesM[thread][i] = mytilesizeM[thread];
  for (long int i=0; i<myntilesN[thread]-1L; i++) mytilesizesN[thread][i] = mytilesizeN[thread];
  for (long int i=0; i<myntilesK[thread]-1L; i++) mytilesizesK[thread][i] = mytilesizeK[thread];
  mytilesizesM[thread][myntilesM[thread]-1L] = mylasttileM[thread];
  mytilesizesN[thread][myntilesN[thread]-1L] = mylasttileN[thread];
  mytilesizesK[thread][myntilesK[thread]-1L] = mylasttileK[thread];


}
void GPUHelper::Tiling(long int mem1,long int mem2,long int m,long int n,long int k){

  // first tile according to how much space is on gpu
  tilesizeN = n;
  tilesizeM = m;
  tilesizeK = k;
  ntilesM=ntilesN=ntilesK=1L;

  while(tilesizeN*tilesizeM+tilesizeK*(tilesizeN+tilesizeM)>mem1){
     if (ntilesN*ntilesM<num_gpus){
        if (tilesizeN>tilesizeM){
           ntilesN++;
           tilesizeN = n/ntilesN;
           if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
        }
        else{
           ntilesM++;
           tilesizeM = m/ntilesM;
           if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
        }
     }
     else{
        if (tilesizeN>tilesizeM){
           if (tilesizeN>tilesizeK){
              ntilesN++;
              tilesizeN = n/ntilesN;
              if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
           }
           else{
              ntilesK++;
              tilesizeK = k/ntilesK;
              if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
           }
        }
        else{
           if (tilesizeM>tilesizeK){
              ntilesM++;
              tilesizeM = m/ntilesM;
              if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
           }
           else{
              ntilesK++;
              tilesizeK = k/ntilesK;
              if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
           }
        }
     }
  }

  // ensure each block of A, B, and C will fit in the temporary CPU buffer
  while(tilesizeN*tilesizeM>mem2){
     if (ntilesN*ntilesM<num_gpus){
        if (tilesizeN>tilesizeM){
           //ntilesN++;
           ntilesN++;
           tilesizeN = n/ntilesN;
           if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
        }
        else{
           ntilesM++;
           ntilesM++;
           tilesizeM = m/ntilesM;
           if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
        }
     }
     else{
        if (tilesizeN>tilesizeM){
           ntilesN++;
           tilesizeN = n/ntilesN;
           if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
        }
        else{
           ntilesM++;
           tilesizeM = m/ntilesM;
           if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
        }
     }
  }

  while(tilesizeN*tilesizeK>mem2){
     if (ntilesN*ntilesM<num_gpus){
        //ntilesN++;
        ntilesN++;
        tilesizeN = n/ntilesN;
        if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
     }
     else{
        if (tilesizeN>tilesizeK){
           ntilesN++;
           tilesizeN = n/ntilesN;
           if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
        }
        else{
           ntilesK++;
           tilesizeK = k/ntilesK;
           if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
        }
     }
  }
  while(tilesizeK*tilesizeM>mem2){
     if (ntilesN*ntilesM<num_gpus){
        ntilesM++;
        //ntilesM++;
        tilesizeM = m/ntilesM;
        if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
     }
     else{
        if (tilesizeK>tilesizeM){
           ntilesK++;
           tilesizeK = k/ntilesK;
           if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
        }
        else{
           ntilesM++;
           tilesizeM = m/ntilesM;
           if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
        }
     }
  }
  // finally make sure that we've tiled enough so each gpu has something to do
  // also, make sure we're load balanced - each GPU has the same work to do
  while(ntilesN*ntilesM<num_gpus && (num_gpus % (ntilesN*ntilesM)) == 0){
     if (tilesizeN>tilesizeM){
        ntilesN++;
        tilesizeN = n/ntilesN;
        if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
     }
     else{
        ntilesM++;
        tilesizeM = m/ntilesM;
        if (m/ntilesM<(double)m/ntilesM) tilesizeM++;
     }
  }

  // double tiling in K so we can pipeline communication/computation
  //ntilesN *= 2;
  //tilesizeN = n/ntilesN;
  //if (n/ntilesN<(double)n/ntilesN) tilesizeN++;
  //ntilesM *= 2;
  //tilesizeM = m/ntilesM;
  //if (m/ntilesM<(double)m/ntilesM) tilesizeM++;


//AED - something is wrong with the tiling ... 
  if (ntilesK < 4)  {
      ntilesK = 8;
      tilesizeK = k/ntilesK;
      if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
  }
    else{
      ntilesK *= 2;
      tilesizeK = k/ntilesK;
      if (k/ntilesK<(double)k/ntilesK) tilesizeK++;
  }
//AED

  lasttileN = n - (ntilesN-1L)*tilesizeN;
  lasttileM = m - (ntilesM-1L)*tilesizeM;
  lasttileK = k - (ntilesK-1L)*tilesizeK;

  tilesizesM = (long int*)malloc(ntilesM*sizeof(long int));
  tilesizesN = (long int*)malloc(ntilesN*sizeof(long int));
  tilesizesK = (long int*)malloc(ntilesK*sizeof(long int));
  for (long int i=0; i<ntilesM-1L; i++) tilesizesM[i] = tilesizeM;
  for (long int i=0; i<ntilesN-1L; i++) tilesizesN[i] = tilesizeN;
  for (long int i=0; i<ntilesK-1L; i++) tilesizesK[i] = tilesizeK;
  tilesizesM[ntilesM-1L] = lasttileM;
  tilesizesN[ntilesN-1L] = lasttileN;
  tilesizesK[ntilesK-1L] = lasttileK;
}
void GPUHelper::TilingWithCpuStealing(long int mem1,long int mem2,long int m,long int n,long int k){
  // compute normal tiling
  Tiling(mem1,mem2,m,n,k);
  // take a slice of the larger of m or n for the cpu
  
  // first let's just try taking a sliver of the last tile of N...
  ntilesNprime = num_cpus;
  ntilesMprime = num_cpus;
  tilesizesNprime = (long int*)malloc(ntilesNprime*sizeof(long int));
  tilesizesMprime = (long int*)malloc(ntilesMprime*sizeof(long int));

  // which dimension will cpu work on?
  if (tilesizeN>tilesizeM){
     // assume the gpu is ~30x faster than a single core:
     if (tilesizeN<30){
        StolenDimension = ' ';
        return;
     }
     tilesizeNprime = tilesizeN/30;

     StolenDimension = 'N';

     // need to figure out new tiles in N (and might as well make them even)
     // TODO: should make these multiples of the warp size, too
     long int newn = n-num_cpus*tilesizeNprime;
     tilesizeN = newn/ntilesN-1;
     lasttileN = tilesizeN;
     for (long int i=0; i<ntilesN; i++)
         tilesizesN[i] = tilesizeN;

     // redo Nprime's numbers
     ntilesNprime   = num_cpus;
     NprimeOffSet   = ntilesN*tilesizeN;
     tilesizeNprime = (n - NprimeOffSet)/ntilesNprime;
     if (tilesizeNprime*ntilesNprime<(n-NprimeOffSet)) tilesizeNprime++;
     lasttileNprime = (n - NprimeOffSet)-(ntilesNprime-1)*tilesizeNprime;

     for (long int i=0; i<ntilesNprime-1; i++) tilesizesNprime[i] = tilesizeNprime;
     tilesizesNprime[ntilesNprime-1] = lasttileNprime;

     // set this just for memory mapping
     lasttileMprime = 0;
     tilesizesMprime[0] = lasttileMprime;
     tilesizeMprime = tilesizeM;
  }
  // do M instead:
  else{
     // assume the gpu is ~30x faster than a single core:
     if (tilesizeM<30){
        StolenDimension = ' ';
        return;
     }
     tilesizeMprime = tilesizeM/30;

     StolenDimension = 'M';

     // need to figure out new tiles in N (and might as well make them even)
     // TODO: should make these multiples of the warp size, too
     long int newm = m-num_cpus*tilesizeMprime;
     tilesizeM = newm/ntilesM-1;
     lasttileM = tilesizeM;
     for (long int i=0; i<ntilesM; i++)
         tilesizesM[i] = tilesizeM;

     // redo Mprime's numbers
     ntilesMprime   = num_cpus;
     MprimeOffSet   = ntilesM*tilesizeM;
     tilesizeMprime = (m - MprimeOffSet)/ntilesMprime;
     if (tilesizeMprime*ntilesMprime<(m-MprimeOffSet)) tilesizeMprime++;
     lasttileMprime = (m - MprimeOffSet)-(ntilesMprime-1)*tilesizeMprime;

     for (long int i=0; i<ntilesMprime-1; i++) tilesizesMprime[i] = tilesizeMprime;
     tilesizesMprime[ntilesMprime-1] = lasttileMprime;

     // set this just for memory mapping
     lasttileNprime = 0;
     tilesizesNprime[0] = lasttileNprime;
     tilesizeNprime = tilesizeN;

     //printf("hey the tile is %5li (%5li) %5li %5li\n",tilesizeMprime,m,ntilesM,tilesizeM);fflush(stdout);
  }
}

void GPUHelper::DGEMM_Timings() {
    long int m,n,k;
    m = n = k = 20000;
    double * A = (double*)malloc(m*k*sizeof(double));
    double * B = (double*)malloc(n*k*sizeof(double));
    double * C = (double*)malloc(m*n*sizeof(double));
    memset((void*)A,'\0',m*k*sizeof(double));
    memset((void*)B,'\0',n*k*sizeof(double));
    memset((void*)C,'\0',m*n*sizeof(double));

    printf("begin tn:\n");
    m = n = k = 10;
    double start = omp_get_wtime();
    GPUTiledDGEMM('t','n',m,n,k,1.0,A,m,B,n,0.0,C,m);
    hipDeviceSynchronize();
    double end = omp_get_wtime();
    printf("%5i %20.12lf\n",10,m*n*k*2.0/(end-start)/1024./1024./1024.);
    fflush(stdout);
    for (long int i = 1; i < 81; i++) {
        m = n = k = 250 * i;
        double start = omp_get_wtime();
        GPUTiledDGEMM('t','n',m,n,k,1.0,A,m,B,n,0.0,C,m);
        hipDeviceSynchronize();
        double end = omp_get_wtime();
        printf("%5li %20.12lf\n",250*i,m*n*k*2.0/(end-start)/1024./1024./1024.);
        fflush(stdout);
    }
    printf("begin nt:\n");
    m = n = k = 10;
    start = omp_get_wtime();
    GPUTiledDGEMM('n','t',m,n,k,1.0,A,m,B,n,0.0,C,m);
    hipDeviceSynchronize();
    end = omp_get_wtime();
    printf("%5i %20.12lf\n",10,m*n*k*2.0/(end-start)/1024./1024./1024.);
    fflush(stdout);
    for (long int i = 1; i < 81; i++) {
        m = n = k = 250 * i;
        double start = omp_get_wtime();
        GPUTiledDGEMM('n','t',m,n,k,1.0,A,m,B,n,0.0,C,m);
        hipDeviceSynchronize();
        double end = omp_get_wtime();
        printf("%5li %20.12lf\n",250*i,m*n*k*2.0/(end-start)/1024./1024./1024.);
        fflush(stdout);
    }
    printf("begin tt:\n");
    m = n = k = 10;
    start = omp_get_wtime();
    GPUTiledDGEMM('t','t',m,n,k,1.0,A,m,B,n,0.0,C,m);
    hipDeviceSynchronize();
    end = omp_get_wtime();
    printf("%5i %20.12lf\n",10,m*n*k*2.0/(end-start)/1024./1024./1024.);
    fflush(stdout);
    for (long int i = 1; i < 81; i++) {
        m = n = k = 250 * i;
        double start = omp_get_wtime();
        GPUTiledDGEMM('t','t',m,n,k,1.0,A,m,B,n,0.0,C,m);
        hipDeviceSynchronize();
        double end = omp_get_wtime();
        printf("%5li %20.12lf\n",250*i,m*n*k*2.0/(end-start)/1024./1024./1024.);
        fflush(stdout);
    }
    printf("begin nn:\n");
    m = n = k = 10;
    start = omp_get_wtime();
    GPUTiledDGEMM('n','n',m,n,k,1.0,A,m,B,n,0.0,C,m);
    hipDeviceSynchronize();
    end = omp_get_wtime();
    printf("%5i %20.12lf\n",10,m*n*k*2.0/(end-start)/1024./1024./1024.);
    fflush(stdout);
    for (long int i = 1; i < 81; i++) {
        m = n = k = 250 * i;
        double start = omp_get_wtime();
        GPUTiledDGEMM('n','n',m,n,k,1.0,A,m,B,n,0.0,C,m);
        hipDeviceSynchronize();
        double end = omp_get_wtime();
        printf("%5li %20.12lf\n",250*i,m*n*k*2.0/(end-start)/1024./1024./1024.);
        fflush(stdout);
    }
    free(A);
    free(B);
    free(C);
    exit(0);
}

}}//end of namespaces


