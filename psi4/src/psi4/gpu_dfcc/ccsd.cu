#include "hip/hip_runtime.h"
/*
 *@BEGIN LICENSE
 *
 * GPU-accelerated density-fitted coupled-cluster, a plugin to:
 *
 * PSI4: an ab initio quantum chemistry software package
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 *
 *@END LICENSE
 */

#include"ccsd.h"
#include"blas.h"
#include<psi4/libmints/matrix.h>
#include<psi4/libmints/vector.h>
#include<psi4/libmints/molecule.h>
#include"gpuhelper.h"
#include<psi4/libmints/mintshelper.h>
#include<psi4/libciomr/libciomr.h>
#include<psi4/libqt/qt.h>
//#include<psi4/libparallel/process.h>
#include<omp.h>


#ifdef HAVE_MKL
    #include<mkl.h>
#else
    #define mkl_set_dynamic(a)
    #define mkl_set_num_threads(a)
    #define mkl_domain_set_num_threads(a,b)
#endif

#define NUMTHREADS 32
#define MAXBLOCKS 65535

__device__ int  GPUKernel_Position(int i,int j) {
  if (i<j){
    return j*(j+1)/2+i;
  }
  return i*(i+1)/2+j;
}
__global__ void GPUKernel_VpVm_tiled(int a, int bstart, int bsize,int v,double * in,double * outp,double * outm) {

    int blockid = blockIdx.x*gridDim.y + blockIdx.y;
    int id      = blockid*blockDim.x + threadIdx.x;

    int v2 = v*v;

    if ( id >= v2*bsize ) return;

    // id : b*v2+c*v+d

    int  d = id%v;
    int  c = (id-d)%(v*v)/v;

    if ( d > c ) return;

    //int  b = (id-d)%(v*bsize)/v;


    //int  c = (id-d-b*v)/(bsize*v);
    int  b = (id-d-c*v)/(v*v);

    if ( b + bstart < a ) return;

    int cd   = c*(c+1)/2 + d;
    int vtri = v*(v+1)/2;
    int bv2  = b*v2;

    //outp[b*vtri+cd] = in[bv2+d*v+c] + in[bv2+c*v+d];
    //outm[b*vtri+cd] = in[bv2+d*v+c] - in[bv2+c*v+d];
    outp[b*vtri+cd] = in[bv2+d*v+c] + in[id];
    outm[b*vtri+cd] = in[bv2+d*v+c] - in[id];
}

__global__ void GPUKernel_VpVm_v2(int a, int b,int v,double * in,double * outp,double * outm) {

    int blockid = blockIdx.x*gridDim.y + blockIdx.y;
    int id      = blockid*blockDim.x + threadIdx.x;

    int v2 = v*v;

    if ( id >= v2 ) return;

    int  d = id%v;
    int  c = (id-d)/v;

    if ( d > c ) return;

    int cd   = GPUKernel_Position(c,d);

    outp[cd] = in[d*v+c] + in[c*v+d];
    outm[cd] = in[d*v+c] - in[c*v+d];
}
__global__ void GPUKernel_VpVm(int a, int v,double * in,double * outp,double * outm) {

    int blockid = blockIdx.x*gridDim.y + blockIdx.y;
    int id      = blockid*blockDim.x + threadIdx.x;

    int v2 = v*v;

    if ( id >= v2*v ) return;

    int  d = id%v;
    int  b = (id-d)%(v2)/v;

    if ( b < a ) return;

    int bma = b - a;

    int  c = (id-d-b*v)/(v2);

    if ( d > c ) return;

    int cd   = GPUKernel_Position(c,d);
    int vtri = v*(v+1)/2;

    outp[bma*vtri+cd] = in[bma*v2+d*v+c] + in[bma*v2+c*v+d];
    outm[bma*vtri+cd] = in[bma*v2+d*v+c] - in[bma*v2+c*v+d];
}
__global__ void GPUKernel_Vm(int a, int v,double * in,double * out) {

    int blockid      = blockIdx.x*gridDim.y + blockIdx.y;
    int id      = blockid*blockDim.x + threadIdx.x;

    if ( id >= v*v*v ) return;

    int  d = id%v;
    int  b = (id-d)%(v*v)/v;
    int  c = (id-d-b*v)/(v*v);

    if ( b < a ) return;
    if ( d > c ) return;

    int cd   = GPUKernel_Position(c,d);
    int vtri = v*(v+1)/2;

    out[(b-a)*vtri+cd] = in[(b-a)*v*v+d*v+c] - in[(b-a)*v*v+c*v+d];
}
__global__ void GPUKernel_Vp(int a, int v,double * in,double * out) {

    int blockid      = blockIdx.x*gridDim.y + blockIdx.y;
    int id      = blockid*blockDim.x + threadIdx.x;

    if ( id >= v*v*v ) return;

    int  d = id%v;
    int  b = (id-d)%(v*v)/v;
    int  c = (id-d-b*v)/(v*v);

    if ( b < a ) return;
    if ( d > c ) return;

    int cd   = GPUKernel_Position(c,d);
    int vtri = v*(v+1)/2;

    out[(b-a)*vtri+cd] = in[(b-a)*v*v+d*v+c] + in[(b-a)*v*v+c*v+d];
}

using namespace psi;


namespace psi{namespace fnocc{

GPUDFCoupledCluster::GPUDFCoupledCluster(std::shared_ptr<Wavefunction> reference_wavefunction, Options &options):
        DFCoupledCluster(reference_wavefunction,options)
{
    common_init();
}

GPUDFCoupledCluster::~GPUDFCoupledCluster()
{
}

// this is where we'll set up cuda/gpu stuff i suppose
void GPUDFCoupledCluster::common_init() {
    /**
      *  GPU helper class knows if we have gpus or not and how to use them.
      *  all gpu memory is allocated by the helper.  
      */
    helper_ = std::shared_ptr<GPUHelper>(new GPUHelper);

    // get device parameters, allocate gpu memory and pinned cpu memory
    helper_->ndoccact = ndoccact;
    helper_->nvirt    = nvirt;
    helper_->nmo      = nmo;
  
    helper_->CudaInit(options_);

    gpubuffer = helper_->gpubuffer;
    left      = helper_->gpumemory / 8.0;
    wasted    = helper_->extraroom / 8.0;
    num_gpus  = helper_->num_gpus;

    long int v = nvirt;
    ngputhreads=NUMTHREADS;
    num=1;
    if ((v*v*v)%ngputhreads==0)
       nblocks = (v*v*v)/ngputhreads;
    else
       nblocks = (v*v*v+ngputhreads-(v*v*v)%ngputhreads)/ngputhreads;
    if (nblocks>MAXBLOCKS){
       num = nblocks/MAXBLOCKS+1;
       nblocks = nblocks/num + 1;
    }

    ncputhreads = omp_get_max_threads();

    if (  options_.get_bool("DGEMM_TIMINGS")  ) {
        helper_->DGEMM_Timings();
    }
  
}

// accumulate results of contraction of (ac|bd) and t2
void GPUDFCoupledCluster::useVabcd1(){

  long int o = ndoccact;
  long int v = nvirt;
  long int oov = o*o*v;
  long int oo  = o*o;
  long int otri = o*(o+1)/2;
  long int vtri = v*(v+1)/2;

  std::shared_ptr<PSIO> psio(new PSIO());

  psio->open(PSIF_DCC_R2,PSIO_OPEN_OLD);
  psio->read_entry(PSIF_DCC_R2,"residual",(char*)&tempv[0],o*o*v*v*sizeof(double));

  // available gpu memory (in doubles)
  long int ndoubles = (left - wasted) - 2*otri*vtri;

  for (long int a = 0; a < v; a++) {

      // do we need to tile loop over b >= a?
      long int ntiles = 1;
      while ( ntiles < v-a )  {
          long int size = (v - a) / ntiles;
          if (size * ntiles < v - a) size++;
          long int max = (size*nQ*v+nQ*v > 2*size*vtri ? size*nQ*v + nQ*v : 2*size*vtri);;

          //if ( ndoubles >= max + 2*size*otri ) break;
          if ( ndoubles >= max + size*nQ*v ) break;
          ntiles++;
      }

      // tile dimensions
      long int * tilesize = (long int *)malloc(ntiles*sizeof(long int));
      for (long int tile = 0; tile < ntiles - 1; tile++) {
          tilesize[tile] = (v-a) / ntiles;
          if ( tilesize[tile] * ntiles < v - a) tilesize[tile]++;
      }
      tilesize[ntiles-1] = (v - a) - tilesize[0] * (ntiles - 1);

      //if (ntiles > 1) printf("%5i/%5i ntiles %5i\n",a,v,ntiles);fflush(stdout);

      for (long int tileb = 0; tileb < ntiles; tileb++) {

          long int bsize = tilesize[tileb];
          long int bstart = a + tileb*tilesize[0];
    
          // contribute to residual
          #pragma omp parallel for schedule (static)
          for (long int ij = 0; ij < o*o; ij++) {
              long int j = ij % o;
              long int i = ( ij - j ) / o;
              int sg     = ( i > j ) ? 1 : -1;
              for (long int b = bstart; b < bstart + bsize; b++) {
                  tempv[a*oo*v+b*oo+i*o+j]    +=    tempr[Position(i,j) * vtri + Position(a,b)]
                                               + sg*tempr[Position(i,j) * vtri + Position(a,b) + otri*vtri];
                  if (a!=b) {
                     tempv[b*oov+a*oo+i*o+j]  +=    tempr[Position(i,j) * vtri + Position(a,b)]
                                               - sg*tempr[Position(i,j) * vtri + Position(a,b) + otri*vtri];
                  }
              }
          }
//gohere
      }
      free(tilesize);

  }

  // contribute to residual
  psio->write_entry(PSIF_DCC_R2,"residual",(char*)&tempv[0],o*o*v*v*sizeof(double));
  psio->close(PSIF_DCC_R2,1);
}

void GPUDFCoupledCluster::Vabcd1(){
  long int o = ndoccact;
  long int v = nvirt;
  long int oov = o*o*v;
  long int oo  = o*o;
  long int otri = o*(o+1)/2;
  long int vtri = v*(v+1)/2;

  std::shared_ptr<PSIO> psio(new PSIO());

  #pragma omp parallel for schedule (static) num_threads(num_gpus)
  for (long int i=0; i<o; i++){
      for (long int j=i; j<o; j++){
          long int ij = Position(i,j);
          for (long int a=0; a<v; a++){
              for (long int b=a; b<v; b++){
                  tempr[ij*vtri+Position(a,b)] =
                     (tb[a*oov+b*oo+i*o+j]+tb[b*oov+a*oo+i*o+j]);
                  tempr[ij*vtri+Position(a,b)+vtri*otri] =
                     (tb[a*oov+b*oo+i*o+j]-tb[b*oov+a*oo+i*o+j]);
              }
              tempr[ij*vtri+Position(a,a)] = tb[a*oov+a*oo+i*o+j];
          }
      }
  }

  if ( v > nQ ) {
      throw PsiException("GPU DFCC will break if Nv > Naux",__FILE__,__LINE__);
  }

  // available gpu memory (in doubles)
  long int ndoubles = (left - wasted) - 2*otri*vtri;
  long int ntiles_ij = 1;

  // do we need to tile ij?
  if ( ndoubles < 0 ) {
      while ( ntiles_ij < otri ) {
          ntiles_ij++;
          long int size  = otri / ntiles_ij;
          if ( size * ntiles_ij < otri ) size++;
          if ( left - wasted - size * 2*vtri ) {
              ndoubles = (left - wasted) - size * 2*vtri;
              break;
          }
      }
      outfile->Printf("    <<< warning >>> tiling composite ij index (%5li tiles)\n",ntiles_ij);
  }
  // sizes of ij tiles:
  long int * tilesize_ij = (long int *)malloc(ntiles_ij*sizeof(long int));
  for (long int tile = 0; tile < ntiles_ij - 1; tile++) {
      tilesize_ij[tile] = otri / ntiles_ij;
      if ( tilesize_ij[tile] * ntiles_ij < otri ) tilesize_ij[tile]++;
  }
  tilesize_ij[ntiles_ij-1] = otri - tilesize_ij[0] * (ntiles_ij - 1);

  for (long int tile_ij = 0; tile_ij < ntiles_ij; tile_ij++) {

      // copy this tile of t2 to the gpus
      #pragma omp parallel for schedule (static) num_threads(num_gpus)
      for (int i = 0; i < num_gpus; i++) {
          int thread = omp_get_thread_num();
          hipSetDevice(thread);
          double * gput2 = gpubuffer[thread];
          hipMemcpy(gput2,                    tempr + tile_ij * tilesize_ij[0] * vtri,              sizeof(double) * tilesize_ij[tile_ij] * vtri,hipMemcpyHostToDevice);
          hipMemcpy(gput2+tilesize_ij[0]*vtri,tempr + tile_ij * tilesize_ij[0] * vtri + otri * vtri,sizeof(double) * tilesize_ij[tile_ij] * vtri,hipMemcpyHostToDevice);
      }

      last_a = v;
      // parallelize over multiple gpus
      #pragma omp parallel for schedule (dynamic) num_threads(num_gpus)
      for (long int a = 0; a < v; a++) {

          if (cpudone && last_a == v) { last_a = a; }

          if (last_a == v) {

          hipStream_t stream;
          hipEvent_t estart,estop;
          hipEventCreate(&estart);
          hipEventCreate(&estop);
          int thread = omp_get_thread_num();
          hipSetDevice(thread);
          double * gput2 = gpubuffer[thread];

          // do we need to tile loop over b >= a?
          long int ntiles = 1;
          while ( ntiles < v-a )  {
              long int size = (v - a) / ntiles;
              if (size * ntiles < v - a) size++;
              long int max = (size*nQ*v+nQ*v > 2*size*vtri ? size*nQ*v + nQ*v : 2*size*vtri);

              //if ( ndoubles >= max + 2*size*otri ) break;
              if ( ndoubles >= max + size*nQ*v ) break;
              ntiles++;
          }

          // tile dimensions
          long int * tilesize = (long int *)malloc(ntiles*sizeof(long int));
          for (long int tile = 0; tile < ntiles - 1; tile++) {
              tilesize[tile] = (v-a) / ntiles;
              if ( tilesize[tile] * ntiles < v - a) tilesize[tile]++;
          }
          tilesize[ntiles-1] = (v - a) - tilesize[0] * (ntiles - 1);

          if (ntiles > 1) outfile->Printf("%5i/%5i ntiles %5i tilesize %5i\n",a,v,ntiles,tilesize[0]);fflush(stdout);

          for (long int tileb = 0; tileb < ntiles; tileb++) {

              long int bsize = tilesize[tileb];
              long int bstart = a + tileb*tilesize[0];

              // shift other buffers by 2 * tilesize_ij * vtri
              long int shift = 2L * tilesize_ij[0] * vtri;

              double * gpuVcdb = gpubuffer[thread] + shift + (bsize*nQ*v + nQ*v > 2*bsize*vtri ? bsize*nQ*v + nQ*v : 2*bsize*vtri);
              double * gpuVm   = gpubuffer[thread] + shift;
              double * gpuVp   = gpubuffer[thread] + shift + bsize*vtri;
              double * gpuA    = gpubuffer[thread] + shift + 2*bsize*vtri;
              double * gpuIqd  = gpubuffer[thread] + shift;
              double * gpuIqc  = gpubuffer[thread] + shift + bsize*nQ*v;

              long int num     = 1;
              long int nblocks = ( bsize*v*v )/ NUMTHREADS;
              if ( (bsize*v*v) % NUMTHREADS != 0 ) {
                 nblocks = (bsize*v*v+NUMTHREADS-(bsize*v*v)%NUMTHREADS)/NUMTHREADS;
              }
              if (nblocks > MAXBLOCKS){
                 num     = nblocks / MAXBLOCKS + 1;
                 nblocks = nblocks / num + 1;
              }

              dim3 dimgrid (nblocks,num);

              stream = NULL;
        
              double start2 = omp_get_wtime();
              //hipDeviceSynchronize();
              //helper_->Check_CUDA_Error(outfile,"before anything. ");
              hipEventRecord(estart,stream);
        
                  hipMemcpyAsync(gpuIqc,Qvv+a*nQ*v,sizeof(double)*nQ*v,hipMemcpyHostToDevice,stream);
                  //hipDeviceSynchronize();
                  //helper_->Check_CUDA_Error(outfile,"memcpy 1");
                  hipMemcpyAsync(gpuIqd,Qvv+bstart*nQ*v,sizeof(double)*bsize*nQ*v,hipMemcpyHostToDevice,stream);
                  //hipDeviceSynchronize();
                  //helper_->Check_CUDA_Error(outfile,"memcpy 2");
                  hipblasDgemm('t','n',v,bsize*v,nQ,1.0,gpuIqc,nQ,gpuIqd,nQ,0.0,gpuVcdb,v);
                  //hipDeviceSynchronize();
                  //helper_->Check_CUDA_Error(outfile,"building v");
       
                  GPUKernel_VpVm_tiled<<<dimgrid,NUMTHREADS>>>(a,bstart,bsize,v,gpuVcdb,gpuVp,gpuVm);
                  //hipDeviceSynchronize();
                  //helper_->Check_CUDA_Error(outfile,"building v+/v-");

                  hipblasDgemm('t','n',tilesize_ij[tile_ij],bsize,vtri,0.5,gput2,                    vtri,gpuVp,vtri,0.0,gpuA,                           tilesize_ij[tile_ij]);
                  hipblasDgemm('t','n',tilesize_ij[tile_ij],bsize,vtri,0.5,gput2+tilesize_ij[0]*vtri,vtri,gpuVm,vtri,0.0,gpuA+bsize*tilesize_ij[tile_ij],tilesize_ij[tile_ij]);

                  hipMemcpyAsync(tempr2[thread],gpuA,sizeof(double)*2*bsize*tilesize_ij[tile_ij],hipMemcpyDeviceToHost,stream);

              hipEventRecord(estop,stream);
        
              while( hipEventQuery(estop) == hipErrorNotReady );
              double end2 = omp_get_wtime();
              for (int ij = 0; ij < tilesize_ij[tile_ij]; ij++) {
                  for (int b = bstart; b < bstart + bsize; b++) {
                      tempr[(ij+tile_ij*tilesize_ij[0])*vtri + Position(a,b)]           = tempr2[thread][(b-bstart)*tilesize_ij[tile_ij]+ij];
                      tempr[(ij+tile_ij*tilesize_ij[0])*vtri + Position(a,b)+otri*vtri] = tempr2[thread][(b-bstart)*tilesize_ij[tile_ij]+ij+bsize*tilesize_ij[tile_ij]];
                  }
              }
//gohere
          }
          free(tilesize);

          }

      }
  }
  free(tilesize_ij);

}
void GPUDFCoupledCluster::FinishVabcd1(){
  long int o = ndoccact;
  long int v = nvirt;
  long int oov = o*o*v;
  long int oo  = o*o;
  long int otri = o*(o+1)/2;
  long int vtri = v*(v+1)/2;

  std::shared_ptr<PSIO> psio(new PSIO());

  // need to build t2+/- for CPU to use
  #pragma omp parallel for schedule (static) num_threads(num_gpus)
  for (long int i=0; i<o; i++){
      for (long int j=i; j<o; j++){
          long int ij = Position(i,j);
          for (long int a=0; a<v; a++){
              for (long int b=a; b<v; b++){
                  tempt[ij*vtri+Position(a,b)] =
                     (tb[a*oov+b*oo+i*o+j]+tb[b*oov+a*oo+i*o+j]);
                  tempt[ij*vtri+Position(a,b)+vtri*otri] =
                     (tb[a*oov+b*oo+i*o+j]-tb[b*oov+a*oo+i*o+j]);
              }
              tempt[ij*vtri+Position(a,a)] = tb[a*oov+a*oo+i*o+j];
          }
      }
  }

  // available gpu memory (in doubles)
  long int ndoubles = (left - wasted) - 2*otri*vtri;
  long int ntiles_ij = 1;


  // available cpu memory (in doubles)
  long int nQmax = nQ > nQ_scf ? nQ : nQ_scf;
  long int dim = 2L*v*v*v;
  if (2*nQmax*o*v>dim)   dim = 2*nQmax*o*v;
  if (o*o*v*v>dim)       dim = o*o*v*v;
  if (nQmax*v*v>dim)     dim = nQmax*v*v;
  if (nQmax*nso*nso>dim) dim = nQmax*nso*nso;

  // do we need to tile ij?
  if ( ndoubles < 0 ) {
      while ( ntiles_ij < otri ) {
          ntiles_ij++;
          long int size  = otri / ntiles_ij;
          if ( size * ntiles_ij < otri ) size++;
          if ( left - wasted - size * 2*vtri ) {
              ndoubles = (left - wasted) - size * 2*vtri;
              break;
          }
      }
      //outfile->Printf("    <<< warning >>> tiling composite ij index (%5li tiles)\n",ntiles_ij);
      //outfile->Printf("    <<< warning >>> tiling composite ij index (%5li tiles)\n",ntiles_ij);
      throw PsiException("  <<< warning >>> tiling composite ij index ... feature temporarily disabled",__FILE__,__LINE__);
  }
  // sizes of ij tiles:
  long int * tilesize_ij = (long int *)malloc(ntiles_ij*sizeof(long int));
  for (long int tile = 0; tile < ntiles_ij - 1; tile++) {
      tilesize_ij[tile] = otri / ntiles_ij;
      if ( tilesize_ij[tile] * ntiles_ij < otri ) tilesize_ij[tile]++;
  }
  tilesize_ij[ntiles_ij-1] = otri - tilesize_ij[0] * (ntiles_ij - 1);


  omp_set_nested(1);
  omp_set_dynamic(0);
  mkl_set_dynamic(0);
  int nthreads = omp_get_max_threads();
  for (long int tile_ij = 0; tile_ij < ntiles_ij; tile_ij++) {

      // copy this tile of t2 to the gpus (already there)

      // parallelize over multiple gpus
      #pragma omp parallel for schedule (dynamic) num_threads(num_gpus + 1)
      for (long int a = last_a; a < v; a++) {

          int thread = omp_get_thread_num();

          if ( thread < num_gpus ) {
              hipStream_t stream;
              hipEvent_t estart,estop;
              hipEventCreate(&estart);
              hipEventCreate(&estop);
              hipSetDevice(thread);
              double * gput2 = gpubuffer[thread];

              // do we need to tile loop over b >= a?
              long int ntiles = 1;
              while ( ntiles < v-a )  {
                  long int size = (v - a) / ntiles;
                  if (size * ntiles < v - a) size++;
                  long int max = (size*nQ*v+nQ*v > 2*size*vtri ? size*nQ*v + nQ*v : 2*size*vtri);

                  //if ( ndoubles >= max + 2*size*otri ) break;
                  if ( ndoubles >= max + size*nQ*v ) break;
                  ntiles++;
              }

              // tile dimensions
              long int * tilesize = (long int *)malloc(ntiles*sizeof(long int));
              for (long int tile = 0; tile < ntiles - 1; tile++) {
                  tilesize[tile] = (v-a) / ntiles;
                  if ( tilesize[tile] * ntiles < v - a) tilesize[tile]++;
              }
              tilesize[ntiles-1] = (v - a) - tilesize[0] * (ntiles - 1);


              for (long int tileb = 0; tileb < ntiles; tileb++) {

                  long int bsize = tilesize[tileb];
                  long int bstart = a + tileb*tilesize[0];

                  // shift other buffers by 2 * tilesize_ij * vtri
                  long int shift = 2L * tilesize_ij[0] * vtri;

                  double * gpuVcdb = gpubuffer[thread] + shift + (bsize*nQ*v + nQ*v > 2*bsize*vtri ? bsize*nQ*v + nQ*v : 2*bsize*vtri);
                  double * gpuVm   = gpubuffer[thread] + shift;
                  double * gpuVp   = gpubuffer[thread] + shift + bsize*vtri;
                  double * gpuA    = gpubuffer[thread] + shift + 2*bsize*vtri;
                  double * gpuIqd  = gpubuffer[thread] + shift;
                  double * gpuIqc  = gpubuffer[thread] + shift + bsize*nQ*v;

                  long int num     = 1;
                  long int nblocks = ( bsize*v*v )/ NUMTHREADS;
                  if ( (bsize*v*v) % NUMTHREADS != 0 ) {
                     nblocks = (bsize*v*v+NUMTHREADS-(bsize*v*v)%NUMTHREADS)/NUMTHREADS;
                  }
                  if (nblocks > MAXBLOCKS){
                     num     = nblocks / MAXBLOCKS + 1;
                     nblocks = nblocks / num + 1;
                  }

                  dim3 dimgrid (nblocks,num);

                  stream = NULL;
        
                  double start2 = omp_get_wtime();
                  //hipDeviceSynchronize();
                  //helper_->Check_CUDA_Error(outfile,"before anything. ");
                  hipEventRecord(estart,stream);
        
                      hipMemcpyAsync(gpuIqc,Qvv+a*nQ*v,sizeof(double)*nQ*v,hipMemcpyHostToDevice,stream);
                      //hipDeviceSynchronize();
                      //helper_->Check_CUDA_Error(outfile,"memcpy 1");
                      hipMemcpyAsync(gpuIqd,Qvv+bstart*nQ*v,sizeof(double)*bsize*nQ*v,hipMemcpyHostToDevice,stream);
                      //hipDeviceSynchronize();
                      //helper_->Check_CUDA_Error(outfile,"memcpy 2");
                      hipblasDgemm('t','n',v,bsize*v,nQ,1.0,gpuIqc,nQ,gpuIqd,nQ,0.0,gpuVcdb,v);
                      //hipDeviceSynchronize();
                      //helper_->Check_CUDA_Error(outfile,"building v");
       
                      GPUKernel_VpVm_tiled<<<dimgrid,NUMTHREADS>>>(a,bstart,bsize,v,gpuVcdb,gpuVp,gpuVm);
                      //hipDeviceSynchronize();
                      //helper_->Check_CUDA_Error(outfile,"building v+/v-");

                      hipblasDgemm('t','n',tilesize_ij[tile_ij],bsize,vtri,0.5,gput2,                    vtri,gpuVp,vtri,0.0,gpuA,                           tilesize_ij[tile_ij]);
                      hipblasDgemm('t','n',tilesize_ij[tile_ij],bsize,vtri,0.5,gput2+tilesize_ij[0]*vtri,vtri,gpuVm,vtri,0.0,gpuA+bsize*tilesize_ij[tile_ij],tilesize_ij[tile_ij]);

                      hipMemcpyAsync(tempr2[thread],gpuA,sizeof(double)*2*bsize*tilesize_ij[tile_ij],hipMemcpyDeviceToHost,stream);

                  hipEventRecord(estop,stream);
        
                  while( hipEventQuery(estop) == hipErrorNotReady );
                  double end2 = omp_get_wtime();
                  for (int ij = 0; ij < tilesize_ij[tile_ij]; ij++) {
                      for (int b = bstart; b < bstart + bsize; b++) {
                          tempr[(ij+tile_ij*tilesize_ij[0])*vtri + Position(a,b)]           = tempr2[thread][(b-bstart)*tilesize_ij[tile_ij]+ij];
                          tempr[(ij+tile_ij*tilesize_ij[0])*vtri + Position(a,b)+otri*vtri] = tempr2[thread][(b-bstart)*tilesize_ij[tile_ij]+ij+bsize*tilesize_ij[tile_ij]];
                      }
                  }
              }
              free(tilesize);

          }else {

              // cpu work

              mkl_set_num_threads(nthreads - num_gpus);

              // do we need to tile loop over b >= a?
              long int ntiles = 1;
/*
              while ( ntiles < v-a )  {
                  long int size = (v - a) / ntiles;
                  if (size * ntiles < v - a) size++;
                  long int max = (size*nQ*v+nQ*v > 2*size*vtri ? size*nQ*v + nQ*v : 2*size*vtri);

                  //if ( ndoubles >= max + 2*size*otri ) break;
                  if ( ndoubles_cpu >= max + size*nQ*v ) break;
                  ntiles++;
              }
*/

              // tile dimensions
              long int * tilesize = (long int *)malloc(ntiles*sizeof(long int));
              for (long int tile = 0; tile < ntiles - 1; tile++) {
                  tilesize[tile] = (v-a) / ntiles;
                  if ( tilesize[tile] * ntiles < v - a) tilesize[tile]++;
              }
              tilesize[ntiles-1] = (v - a) - tilesize[0] * (ntiles - 1);

              if (ntiles > 1) outfile->Printf("%5i/%5i ntiles %5i tilesize %5i (cpu) \n",a,v,ntiles,tilesize[0]);fflush(stdout);

              for (long int tileb = 0; tileb < ntiles; tileb++) {

                  long int bsize = tilesize[tileb];
                  long int bstart = a + tileb*tilesize[0];

                  // shift other buffers by 2 * tilesize_ij * vtri
                  long int shift = 0;//2L * tilesize_ij[0] * vtri;

                  double * gpuVm   = integrals + shift;
                  double * gpuVp   = integrals + shift + bsize*vtri;
                  double * gpuA    = integrals + shift + 2*bsize*vtri;
                  double * gpuVcdb = integrals + shift + 3*bsize*vtri;//(bsize*nQ*v + nQ*v > 2*bsize*vtri ? bsize*nQ*v + nQ*v : 2*bsize*vtri);
                  //double * gpuIqd  = integrals + shift;
                  //double * gpuIqc  = integrals + shift + bsize*nQ*v;

                  double start2 = omp_get_wtime();
        
                  //hipMemcpyAsync(gpuIqc,Qvv+a*nQ*v,sizeof(double)*nQ*v,hipMemcpyHostToDevice,stream);
                  //hipMemcpyAsync(gpuIqd,Qvv+bstart*nQ*v,sizeof(double)*bsize*nQ*v,hipMemcpyHostToDevice,stream);
                  F_DGEMM('t','n',v,bsize*v,nQ,1.0,Qvv+a*nQ*v,nQ,Qvv+bstart*nQ*v,nQ,0.0,gpuVcdb,v);
       
                  #pragma omp parallel for schedule (dynamic) num_threads(nthreads - num_gpus)
                  for (int d = 0; d < v; d++) {
                      for (int c = d; c < v; c++) {
                          int cd   = c*(c+1)/2 + d;
                          for (int b = bstart; b < v; b++) {
                              int id   = d + c*v + (b-bstart)*v*v;
                              int bv2  = (b-bstart)*v*v;
                               gpuVp[(b-bstart)*vtri+cd] = gpuVcdb[bv2+d*v+c] + gpuVcdb[id];
                               gpuVm[(b-bstart)*vtri+cd] = gpuVcdb[bv2+d*v+c] - gpuVcdb[id];
                          }
                      }
                  }

                  F_DGEMM('t','n',tilesize_ij[tile_ij],bsize,vtri,0.5,tempt,                    vtri,gpuVp,vtri,0.0,gpuA,                           tilesize_ij[tile_ij]);
                  F_DGEMM('t','n',tilesize_ij[tile_ij],bsize,vtri,0.5,tempt+tilesize_ij[0]*vtri,vtri,gpuVm,vtri,0.0,gpuA+bsize*tilesize_ij[tile_ij],tilesize_ij[tile_ij]);

                  //hipMemcpyAsync(tempr2[thread],gpuA,sizeof(double)*2*bsize*tilesize_ij[tile_ij],hipMemcpyDeviceToHost,stream);
        
                  #pragma omp parallel for schedule (dynamic) num_threads(nthreads - num_gpus)
                  for (int ij = 0; ij < tilesize_ij[tile_ij]; ij++) {
                      for (int b = bstart; b < bstart + bsize; b++) {
                          tempr[(ij+tile_ij*tilesize_ij[0])*vtri + Position(a,b)]           = gpuA[(b-bstart)*tilesize_ij[tile_ij]+ij];
                          tempr[(ij+tile_ij*tilesize_ij[0])*vtri + Position(a,b)+otri*vtri] = gpuA[(b-bstart)*tilesize_ij[tile_ij]+ij+bsize*tilesize_ij[tile_ij]];
                      }
                  }
              }
              free(tilesize);

          }
      }
  }
  free(tilesize_ij);

  omp_set_nested(0);
  omp_set_dynamic(1);
  mkl_set_dynamic(1);
  mkl_set_num_threads(nthreads);

}


void GPUDFCoupledCluster::CudaInit(){
  num_gpus = 0;

  cublasInit();
  helper_->Check_CUDA_Error(stdout,"cudaInit");
  struct hipDeviceProp_t cudaProp;
  int gpu_id;

  // how many GPUs do we have?
  hipGetDeviceCount(&num_gpus);
  helper_->Check_CUDA_Error(stdout,"hipGetDeviceCount");

  if ( num_gpus == 0 ) { 
      throw PsiException("    Error: no cuda capable device detected.",__FILE__,__LINE__);
  }

  if (options_["NUM_GPUS"].has_changed()) {
      num_gpus = options_.get_int("NUM_GPUS");
  }

  hipGetDevice(&gpu_id);
  helper_->Check_CUDA_Error(stdout,"hipGetDevice");
  hipGetDeviceProperties( &cudaProp,gpu_id );
  helper_->Check_CUDA_Error(stdout,"hipGetDeviceProperties");
  outfile->Printf("\n");
  outfile->Printf("  _________________________________________________________\n");
  outfile->Printf("  CUDA device properties:\n");
  outfile->Printf("  name:                 %20s\n",cudaProp.name);
  outfile->Printf("  major version:        %20d\n",cudaProp.major);
  outfile->Printf("  minor version:        %20d\n",cudaProp.minor);
  outfile->Printf("  canMapHostMemory:     %20d\n",cudaProp.canMapHostMemory);
  outfile->Printf("  totalGlobalMem:       %20lu mb\n",cudaProp.totalGlobalMem/(1024*1024));
  outfile->Printf("  sharedMemPerBlock:    %20lu\n",cudaProp.sharedMemPerBlock);
  outfile->Printf("  clockRate:            %20.3f ghz\n",cudaProp.clockRate/1.0e6);
  outfile->Printf("  regsPerBlock:         %20d\n",cudaProp.regsPerBlock);
  outfile->Printf("  warpSize:             %20d\n",cudaProp.warpSize);
  outfile->Printf("  maxThreadsPerBlock:   %20d\n",cudaProp.maxThreadsPerBlock);
  outfile->Printf("  _________________________________________________________\n");
  outfile->Printf("\n");
  //fflush(outfile);

  // device memory left after some arrays (no, now total memory)
  int v = nvirt;
  left = cudaProp.totalGlobalMem/8.;// - 3*o*o*v*v - o*v-nmo*nmo;
  wasted = 200*1024*1024/8.; // leave an extra 200 mb on there.

  ngputhreads=NUMTHREADS;
  num=1;
  if ((v*v*v)%ngputhreads==0)
     nblocks = (v*v*v)/ngputhreads;
  else
     nblocks = (v*v*v+ngputhreads-(v*v*v)%ngputhreads)/ngputhreads;
  if (nblocks>MAXBLOCKS){
     num = nblocks/MAXBLOCKS+1;
     nblocks = nblocks/num + 1;
  }

  hipDeviceReset();
  helper_->Check_CUDA_Error(stdout,"hipDeviceReset");
}

void GPUDFCoupledCluster::CudaFinalize(){

  #pragma omp parallel for schedule (static) num_threads(num_gpus)
  for (int i=0; i<num_gpus; i++){ 
      int thread = omp_get_thread_num();
      hipSetDevice(thread);
      hipFree(gpubuffer[thread]);
  }
  hipDeviceReset();
}
void GPUDFCoupledCluster::AllocateGPUMemory(){

  gpubuffer = (double**)malloc(num_gpus*sizeof(double*));
  #pragma omp parallel for schedule (static) num_threads(num_gpus)
  for (int i=0; i<num_gpus; i++){ 
      int thread = omp_get_thread_num();

      hipSetDevice(thread);
      helper_->Check_CUDA_Error(stdout,"hipSetDevice");

      hipMalloc((void**)&gpubuffer[thread],sizeof(double)*(left-wasted));
      helper_->Check_CUDA_Error(stdout,"gpu memory");

  }

}

void GPUDFCoupledCluster::AllocateMemory() {

  if (nirrep_>1){
     throw PsiException("df_ccsd requires symmetry c1",__FILE__,__LINE__);
  }

  ischolesky_ = ( options_.get_str("DF_BASIS_CC") == "CHOLESKY" );
  nQ          = (int)Process::environment.globals["NAUX (CC)"];
  nQ_scf      = (int)Process::environment.globals["NAUX (SCF)"];

  int count=0;
  eps = (double*)malloc((ndoccact+nvirt)*sizeof(double));
  std::shared_ptr<Vector> eps_test = reference_wavefunction_->epsilon_a();
  for (int h=0; h<nirrep_; h++){
      for (int norb = frzcpi_[h]; norb<doccpi_[h]; norb++){
          eps[count++] = eps_test->get(h,norb);
      }
  }
  for (int h=0; h<nirrep_; h++){
      for (int norb = doccpi_[h]; norb<nmopi_[h]-frzvpi_[h]; norb++){
          eps[count++] = eps_test->get(h,norb);
      }
  }

  long int o = ndoccact;
  long int v = nvirt;

  /*========================================================
     ccsd memory requirements:
    
     tb:     o^2v^2
     tempt:  o^2v^2+ov ( actually o(o+1)v(v+1) + ov )
     tempv:  max (o^2v^2+ov , o*v*nQ)
     integrals: max(2v^3,nQ*nso^2, o^2v^2, 2v^3, 2nQ*o*v) (this is a minimum)
     Abij (SJS v^4 result): o(o+1)v/2
     Sbij (SJS v^4 result): o(o+1)v/2
     other stuff: 2ov+2v^2+(o+v)
    
     total: 3o^2v^2 + 2v^3  + o(o+1)v + 4ov  + 2v^2 + (o+v)  or 
            4o^2v^2         + o(o+1)v + 4ov  + 2v^2 + (o+v)  or
            3o^2v^2 + 2ovnQ + o(o+1)v + 4ov  + 2v^2 + (o+v)

     compare to the requirements for the (T) part:

            2o^2v^2 + 3v^3*nthreads + o^3v + ov
    
  ========================================================*/

  // reduce available memory by the amount required by the helper class
  memory -= helper_->max_mapped_memory;

  long int nQmax = nQ > nQ_scf ? nQ : nQ_scf;

  // for the df version, the dimension of the large buffer:
  long int dim = 2L*v*v*v;
  if (2*nQmax*o*v>dim)   dim = 2*nQmax*o*v;
  if (o*o*v*v>dim)    dim = o*o*v*v;
  if (nQmax*v*v>dim)     dim = nQmax*v*v;
  if (nQmax*nso*nso>dim) dim = nQmax*nso*nso;

  double total_memory = dim+(o*o*v*v+o*v)+(o*(o+1)*v*(v+1)+o*v)+o*o*v*v+2.*o*v+2.*v*v;
  long int max = nvirt*nvirt*nQmax > (nfzv+ndocc+nvirt)*ndocc*nQmax ? nvirt*nvirt*nQmax : (nfzv+ndocc+nvirt)*ndocc*nQmax;
  double df_memory    = nQmax*(o*o+o*v)+max;

  total_memory       *= 8./1024./1024.;
  df_memory          *= 8./1024./1024.;

  outfile->Printf("  Total memory requirements:       %9.2lf mb\n",df_memory+total_memory);
  outfile->Printf("  3-index integrals:               %9.2lf mb\n",df_memory);
  outfile->Printf("  CCSD intermediates:              %9.2lf mb\n",total_memory);
  outfile->Printf("\n");

  if (1.0 * memory / 1024. / 1024. < total_memory + df_memory) {
     outfile->Printf("\n");
     outfile->Printf("  error: not enough memory for ccsd.  increase available memory by %7.2lf mb\n",total_memory+df_memory-1.0*memory/1024./1024.);
     outfile->Printf("\n");
     //fflush(outfile);
     throw PsiException("not enough memory (ccsd).",__FILE__,__LINE__);
  }
  if (options_.get_bool("COMPUTE_TRIPLES")) {
      long int nthreads = omp_get_max_threads();
      double tempmem = 8.*(2L*o*o*v*v+o*o*o*v+o*v+3L*v*v*v*nthreads);
      if (tempmem > memory) {
          outfile->Printf("\n  <<< warning! >>> switched to low-memory (t) algorithm\n\n");
      }
      if (tempmem > memory || options_.get_bool("TRIPLES_LOW_MEMORY")){
         throw PsiException("low-memory triples option not yet implemented",__FILE__,__LINE__);
         //DPG commented out to remove unreachable warning
	 //isLowMemory = true;
         //tempmem = 8.*(2L*o*o*v*v+o*o*o*v+o*v+5L*o*o*o*nthreads);
      }
      outfile->Printf("  memory requirements for CCSD(T): %9.2lf mb\n\n",tempmem/1024./1024.);
  }
  hipHostMalloc((void**)&Qvv,nvirt*nvirt*nQ*sizeof(double));
  hipDeviceSynchronize();
  helper_->Check_CUDA_Error(stdout,"allocate host Qvv");

  tempr = (double*)malloc(o*(o+1)*v*(v+1)/2*sizeof(double));

  hipDeviceSynchronize();
  helper_->Check_CUDA_Error(stdout,"allocate host tempr");

  // o*(o+1)*v mapped memory for each gpu:
  // for now, give the choice of using helper's or allocating more.  TODO:
  // need to figure out a cleaner way to choose the memory we want to pin
  // and Qvv needs to be considerred as well.
  if ( o*(o+1)/v*sizeof(double) < helper_->max_mapped_memory_per_thread ) {
      tempr2 = helper_->tmp;
  }else {
      tempr2 = (double**)malloc(num_gpus*sizeof(double*));
      #pragma omp parallel for schedule (static) num_threads(num_gpus)
      for (long int i=0; i<num_gpus; i++){
          long int thread = 0;
          #ifdef _OPENMP
            thread = omp_get_thread_num();
          #endif
          hipSetDevice(thread);
          helper_->Check_CUDA_Error(stdout,"hipSetDevice");
          hipHostMalloc((void**)&tempr2[thread],o*(o+1)*v*sizeof(double));
          helper_->Check_CUDA_Error(stdout,"cpu tempr2");
      }
  }


  // allocate some memory for 3-index tensors
  Qoo = (double*)malloc(ndoccact*ndoccact*nQmax*sizeof(double));
  Qov = (double*)malloc(ndoccact*nvirt*nQmax*sizeof(double));

  long int                       tempvdim = o*o*v*v+o*v;
  if ( nQmax * o * v > tempvdim) tempvdim = nQmax * o * v;

  integrals = (double*)malloc(dim*sizeof(double));
  tempt     = (double*)malloc((o*(o+1)*v*(v+1)+o*v)*sizeof(double));
  tempv     = (double*)malloc(tempvdim*sizeof(double));
  Abij      = (double*)malloc(o*(o+1)/2*v*sizeof(double));
  Sbij      = (double*)malloc(o*(o+1)/2*v*sizeof(double));
  tb        = (double*)malloc(o*o*v*v*sizeof(double));
  w1        = (double*)malloc(o*v*sizeof(double));
  t1        = (double*)malloc(o*v*sizeof(double));
  I1        = (double*)malloc(v*v*sizeof(double));
  I1p       = (double*)malloc(v*v*sizeof(double));

  memset((void*)integrals,'\0',dim*sizeof(double));
  memset((void*)tempv,'\0',tempvdim*sizeof(double));
  memset((void*)tempt,'\0',(o*(o+1)*v*(v+1)+o*v)*sizeof(double));
  memset((void*)tempr,'\0',(o*(o+1)*v*(v+1)/2)*sizeof(double));
  memset((void*)tb,'\0',o*o*v*v*sizeof(double));
  memset((void*)w1,'\0',o*v*sizeof(double));
  memset((void*)t1,'\0',o*v*sizeof(double));
  memset((void*)I1,'\0',v*v*sizeof(double));
  memset((void*)I1p,'\0',v*v*sizeof(double));
  memset((void*)Abij,'\0',o*(o+1)/2*v*sizeof(double));
  memset((void*)Sbij,'\0',o*(o+1)/2*v*sizeof(double));

  // DIIS:
  diisvec    = (double*)malloc(sizeof(double)*(maxdiis+1));
  memset((void*)diisvec,'\0',(maxdiis+1)*sizeof(double));

  // new 3-index stuff for t1-transformed integrals:
  Fij   = (double*)malloc(o*o*sizeof(double));
  Fia   = (double*)malloc(o*v*sizeof(double));
  Fai   = (double*)malloc(o*v*sizeof(double));
  Fab   = (double*)malloc(v*v*sizeof(double));
  Ca_R  = (double*)malloc(nso*(nmo+nfzc+nfzv)*sizeof(double));
  Ca_L  = (double*)malloc(nso*(nmo+nfzc+nfzv)*sizeof(double));

  Ca = reference_wavefunction_->Ca()->pointer();

  // one-electron integrals
  std::shared_ptr<BasisSet> basis = reference_wavefunction_->basisset();
  std::shared_ptr<MintsHelper> mints(new MintsHelper(basis,options_));
  H = mints->so_kinetic();
  H->add(mints->so_potential());

}

// GPU kernels!
__global__ void GPUKernel_Iqdb(int a,int v,int nQ,double * in,double * out) {

    int blockid = blockIdx.x*gridDim.y + blockIdx.y;
    int id      = blockid*blockDim.x + threadIdx.x;

    if ( id >= v*v*nQ ) return;

    int  q = id%nQ;
    int  d = (id-q)%(nQ*v)/nQ;
    int  b = (id-q-d*nQ)/(nQ*v);

    if ( b < a ) return;

    int id2 = (b-a)*nQ*v+d*nQ+q;
    out[id2] = in[id];

}
typedef struct {
    int id;
    GPUDFCoupledCluster * cc;
} mega;

void *doit(void*x) {

  mega * m = (mega * )x;
  m->cc->pthreadCCResidual(m->id);

  return (NULL);
}

void GPUDFCoupledCluster::pthreadCCResidual(int id) {
    bool timer = options_.get_bool("CC_TIMINGS");
    long int o = ndoccact;
    long int v = nvirt;

        //////// start gpu section! ////////
        if (id==0)
        {
// AED
            omp_set_num_threads(num_gpus);
    
            int nthreads = omp_get_max_threads();

            #pragma omp parallel for schedule (static) num_threads(num_gpus)
            for (int i = 0 ; i < num_gpus; i++) {
                int mythread = omp_get_thread_num();
                hipSetDevice(mythread);
            }
            double start = omp_get_wtime();

            Vabcd1();

            if (last_a == v) {
                gpudone = true;
                if (timer) {
                    outfile->Printf("        A2 =      t(c,d,i,j) (ac|bd)                                    %6.2lf\n",omp_get_wtime()-start);
                }
            }
            else 
                gpudone = false;
// AED
//gpudone = false;
    
        //////// end gpu section! ////////
        } 
        //////// start cpu section! ////////
        else
        {

// AED
///*
            int mythread = omp_get_thread_num();

            // pthread has NO idea what the right number of threads is ...
            int nthreads = ncputhreads;//omp_get_max_threads();

            if (nthreads > 1 + num_gpus) nthreads -= num_gpus;

            omp_set_num_threads(nthreads);
            mkl_set_num_threads(nthreads);
            mkl_domain_set_num_threads(nthreads, MKL_DOMAIN_BLAS);

            double start;

            // C2 = -1/2 t(bc,kj) [ (ki|ac) - 1/2 t(ad,li) (kd|lc) ] 
            //      +    t(bc,ki) [ (kj|ac) - 1/2 t(ad,lj) (kd|lc) ] 
            if (timer) start = omp_get_wtime();

            if (gpudone) helper_->GPUTiledDGEMM('n','t',o*v,o*v,nQ,1.0,Qov,o*v,Qov,o*v,0.0,integrals,o*v);
            else F_DGEMM('n','t',o*v,o*v,nQ,1.0,Qov,o*v,Qov,o*v,0.0,integrals,o*v);

            if (gpudone) {
                omp_set_num_threads(ncputhreads);
                mkl_set_num_threads(ncputhreads);
                mkl_domain_set_num_threads(ncputhreads, MKL_DOMAIN_BLAS);
            }

            //printf("position 1 %20.12lf\n",omp_get_wtime()-start);fflush(stdout);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int a = 0; a < v; a++) {
                for (int i = 0; i < o; i++) {
                    for (int l = 0; l < o; l++) {
                        for (int d = 0; d < v; d++) {
                            tempt[a*o*o*v+i*o*v+l*v+d] = tb[a*o*o*v+d*o*o+l*o+i];
                        }
                    }
                }
            }
            //printf("position 2 %20.12lf\n",omp_get_wtime()-start);fflush(stdout);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int l = 0; l < o; l++) {
                for (int d = 0; d < v; d++) {
                    for (int k = 0; k < o; k++) {
                        for (int c = 0; c < v; c++) {
                            tempv[k*o*v*v+c*o*v+l*v+d] = integrals[k*o*v*v+d*o*v+l*v+c];
                        }
                    }
                }
            }
            // hang out until the gpu finishes ...
//            double wait = omp_get_wtime();
//            double accum = 0.0;
//            do {
//                if ( omp_get_wtime() - wait > 5.0 ) {
//                    accum += omp_get_wtime() - wait;
//                    wait = omp_get_wtime();
//                    outfile->Printf("gpu has taken an extra %6.2lf s\n",accum);
//                }
//            }while(!gpudone);

            //printf("position 3 %20.12lf\n",omp_get_wtime()-start);fflush(stdout);
//            if (gpudone) helper_->GPUTiledDGEMM('t','n',o*v,o*v,o*v,-0.5,tempv,o*v,tempt,o*v,0.0,integrals,o*v);
//            else         F_DGEMM('t','n',o*v,o*v,o*v,-0.5,tempv,o*v,tempt,o*v,0.0,integrals,o*v);
            long int gpuchunk = 0;
            long int odone    = 0;
            for (int i = 0; i < o; i++) {
                if (!gpudone) {
                    F_DGEMM('t','n',o*v,v,o*v,-0.5,tempv,o*v,tempt+i*o*v*v,o*v,0.0,integrals+i*o*v*v,o*v);
                }else {
                    gpuchunk = o - i;
                    odone    = i;
                    break;
                }
            }
            if (gpudone && gpuchunk > 0) {
                helper_->GPUTiledDGEMM('t','n',o*v,gpuchunk*v,o*v,-0.5,tempv,o*v,tempt+odone*o*v*v,o*v,0.0,integrals+odone*o*v*v,o*v);
            }

            if (gpudone) {
                omp_set_num_threads(ncputhreads);
                mkl_set_num_threads(ncputhreads);
                mkl_domain_set_num_threads(ncputhreads, MKL_DOMAIN_BLAS);
            }

            //printf("position 4 %20.12lf\n",omp_get_wtime()-start);fflush(stdout);
            if (gpudone) helper_->GPUTiledDGEMM('t','t',v*v,o*o,nQ,1.0,Qvv,nQ,Qoo,o*o,0.0,tempv,v*v);
            else         F_DGEMM('t','t',v*v,o*o,nQ,1.0,Qvv,nQ,Qoo,o*o,0.0,tempv,v*v);

            if (gpudone) {
                omp_set_num_threads(ncputhreads);
                mkl_set_num_threads(ncputhreads);
                mkl_domain_set_num_threads(ncputhreads, MKL_DOMAIN_BLAS);
            }

            //printf("position 5 %20.12lf\n",omp_get_wtime()-start);fflush(stdout);
            //F_DGEMM('n','t',v*v,o*o,nQ,1.0,Qvv,v*v,Qoo,o*o,0.0,tempv,v*v);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int a = 0; a < v; a++) {
                for (int i = 0; i < o; i++) {
                    for (int k = 0; k < o; k++) {
                        for (int c = 0; c < v; c++) {
                            integrals[a*o*o*v+i*o*v+k*v+c] += tempv[k*o*v*v+i*v*v+a*v+c];
                        }
                    }
                }
            }
            //printf("position 6 %20.12lf\n",omp_get_wtime()-start);fflush(stdout);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int b = 0; b < v; b++) {
                for (int j = 0; j < o; j++) {
                    for (int k = 0; k < o; k++) {
                        for (int c = 0; c < v; c++) {
                            tempt[b*o*o*v+j*o*v+k*v+c] = tb[b*o*o*v+c*o*o+k*o+j];
                        }
                    }
                }
            }
            //printf("position 7 %20.12lf\n",omp_get_wtime()-start);fflush(stdout);
//            if (gpudone) helper_->GPUTiledDGEMM('t','n',o*v,o*v,o*v,-1.0,integrals,o*v,tempt,o*v,0.0,tempv,o*v);
//            else         F_DGEMM('t','n',o*v,o*v,o*v,-1.0,integrals,o*v,tempt,o*v,0.0,tempv,o*v);

            gpuchunk = 0;
            odone    = 0;
            for (int i = 0; i < o; i++) {
                if (!gpudone) {
                    F_DGEMM('t','n',o*v,v,o*v,-1.0,integrals,o*v,tempt+i*o*v*v,o*v,0.0,tempv+i*o*v*v,o*v);
                }else {
                    gpuchunk = o - i;
                    odone    = i;
                    break;
                }
            }
            if (gpudone && gpuchunk > 0) {
                helper_->GPUTiledDGEMM('t','n',o*v,gpuchunk*v,o*v,-1.0,integrals,o*v,tempt+odone*o*v*v,o*v,0.0,tempv+odone*o*v*v,o*v);
            }

            if (gpudone) {
                omp_set_num_threads(ncputhreads);
                mkl_set_num_threads(ncputhreads);
                mkl_domain_set_num_threads(ncputhreads, MKL_DOMAIN_BLAS);
            }

            //printf("position 8 %20.12lf\n",omp_get_wtime()-start);fflush(stdout);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int a = 0; a < v; a++) {
                for (int b = 0; b < v; b++) {
                    for (int i = 0; i < o; i++) {
                        for (int j = 0; j < o; j++) {
                            tempt[a*o*o*v+b*o*o+i*o+j] = 0.5 * tempv[b*o*o*v+j*o*v+a*o+i] + tempv[b*o*o*v+i*o*v+a*o+j];
                        }
                    }
                }
            }
            //printf("position 9 %20.12lf\n",omp_get_wtime()-start);fflush(stdout);

            // first contribution to residual 
            std::shared_ptr<PSIO> psio(new PSIO());
            psio->open(PSIF_DCC_R2,PSIO_OPEN_NEW);
            psio->write_entry(PSIF_DCC_R2,"residual",(char*)&tempt[0],o*o*v*v*sizeof(double));
            psio->close(PSIF_DCC_R2,1);
            //printf("position 10 %20.12lf\n",omp_get_wtime()-start);fflush(stdout);
            if (timer) {
                outfile->Printf("\n");
                outfile->Printf("        C2 = -1/2 t(b,c,k,j) [ (ki|ac) - 1/2 t(a,d,l,i) (kd|lc) ]\n");
                outfile->Printf("                + t(b,c,k,i) [ (kj|ac) - 1/2 t(a,d,l,j) (kd|lc) ]       %6.2lf\n",omp_get_wtime()-start);
                start = omp_get_wtime();
            }

            // now singles residual:

            // D1: F(ai)
            C_DCOPY(o*v,Fai,1,w1,1);

            // A1 (G):  U(c,d,k,l) (ad|kc)
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int d = 0; d < v; d++) {
                for (int i = 0; i < o; i++) {
                    for (int k = 0; k < o; k++) {
                        for (int c = 0; c < v; c++) {
                            tempt[d*o*o*v+i*o*v+k*v+c] = (2.0*tb[c*o*o*v+d*o*o+k*o+i] - tb[c*o*o*v+d*o*o+i*o+k]);
                        }
                    }
                }
            }
            if (gpudone) helper_->GPUTiledDGEMM('t','n',o*v,nQ,o*v,1.0,tempt,o*v,Qov,o*v,0.0,tempv,o*v);
            else F_DGEMM('t','n',o*v,nQ,o*v,1.0,tempt,o*v,Qov,o*v,0.0,tempv,o*v);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int q = 0; q < nQ; q++) {
                for (int a = 0; a < v; a++) {
                    for (int b = 0; b < v; b++) {
                        integrals[q*v*v+b*v+a] = Qvv[a*v*nQ+b*nQ+q];
                    }
                }
            }
            //#pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            //for (int q = 0; q < nQ; q++) {
            //    for (int a = 0; a < v; a++) {
            //        C_DCOPY(v,Qvv+q*v*v+a*v,1,integrals+q*v*v+a,v);
            //    }
            //}
            //if (gpudone) helper_->GPUTiledDGEMM('n','t',o,v,v*nQ,1.0,tempv,o,integrals,v,1.0,w1,o);
            //else F_DGEMM('n','t',o,v,v*nQ,1.0,tempv,o,integrals,v,1.0,w1,o);
            F_DGEMM('n','t',o,v,v*nQ,1.0,tempv,o,integrals,v,1.0,w1,o);

            if (timer) {
                outfile->Printf("        A1 =      U(c,d,k,l) (ad|kc)                                    %6.2lf\n",omp_get_wtime()-start);
                start = omp_get_wtime();
            }

            // B1 (H): -U(a,c,k,l) (ki|lc)
            if (gpudone) helper_->GPUTiledDGEMM('n','t',o*v,o*o,nQ,1.0,Qov,o*v,Qoo,o*o,0.0,integrals,o*v);
            else F_DGEMM('n','t',o*v,o*o,nQ,1.0,Qov,o*v,Qoo,o*o,0.0,integrals,o*v);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int i = 0; i < o; i++) {
                for (int c = 0; c < v; c++) {
                    for (int k = 0; k < o; k++) {
                        for (int l = 0; l < o; l++) {
                            tempv[i*o*o*v+c*o*o+k*o+l] = integrals[k*o*o*v+i*o*v+l*v+c];
                        }
                    }
                }
            }
            C_DCOPY(o*o*v*v,tb,1,tempt,1);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int a = 0; a < v; a++) {
                for (int c = 0; c < v; c++) {
                    for (int k = 0; k < o; k++) {
                        C_DAXPY(o,-0.5,tb+a*o*o*v+c*o*o+k,o,tempt+a*o*o*v+c*o*o+k*o,1);
                    }
                }
            }
            if (gpudone) helper_->GPUTiledDGEMM('t','n',o,v,o*o*v,-2.0,tempv,o*o*v,tempt,o*o*v,1.0,w1,o);
            else F_DGEMM('t','n',o,v,o*o*v,-2.0,tempv,o*o*v,tempt,o*o*v,1.0,w1,o);

            if (timer) {
                outfile->Printf("        B1 =    - U(a,c,k,l) (ki|lc)                                    %6.2lf\n",omp_get_wtime()-start);
                start = omp_get_wtime();
            }

            // C1
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int a = 0; a < v; a++) {
                for (int i = 0; i < o; i++) {
                    double dum = 0.0;
                    for (int k = 0; k < o; k++) {
                        for (int c = 0; c < v; c++) {
                            dum += Fia[k*v+c] * (2.0*tb[a*o*o*v+c*o*o+i*o+k] - tb[a*o*o*v+c*o*o+k*o+i]);
                        }
                    }
                    w1[a*o+i] += dum;
                }
            }

            if (timer) {
                outfile->Printf("        C1 =      F(k,c) U(a,c,i,k)                                     %6.2lf\n",omp_get_wtime()-start);
                start = omp_get_wtime();
            }

            // D2: 1/2 U(b,c,j,k) [ L(a,i,k,c) + 1/2 U(a,d,i,l) L(l,d,k,c) ] 
            if (gpudone) helper_->GPUTiledDGEMM('n','t',o*v,o*v,nQ,1.0,Qov,o*v,Qov,o*v,0.0,integrals,o*v);
            else F_DGEMM('n','t',o*v,o*v,nQ,1.0,Qov,o*v,Qov,o*v,0.0,integrals,o*v);

            if (gpudone) {
                omp_set_num_threads(ncputhreads);
                mkl_set_num_threads(ncputhreads);
                mkl_domain_set_num_threads(ncputhreads, MKL_DOMAIN_BLAS);
            }

            C_DCOPY(o*o*v*v,integrals,1,tempv,1);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int l = 0; l < o; l++) {
                for (int d = 0; d < v; d++) {
                    for (int k = 0; k < o; k++) {
                        for (int c = 0; c < v; c++) {
                            tempv[l*o*v*v+d*o*v+k*v+c] -= 0.5 * integrals[l*o*v*v+c*o*v+k*v+d];
                        }
                    }
                }
            }
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int l = 0; l < o; l++) {
                for (int d = 0; d < v; d++) {
                    for (int a = 0; a < v; a++) {
                        for (int i = 0; i < o; i++) {
                            tempt[a*o*o*v+i*o*v+l*v+d] = 2.0 * tb[a*o*o*v+d*o*o+i*o+l]-tb[a*o*o*v+d*o*o+l*o+i];
                            //tempt[l*o*v*v+d*o*v+a*o+i] = 2.0 * tb[a*o*o*v+d*o*o+i*o+l]-tb[a*o*o*v+d*o*o+l*o+i];
                        }
                    }
                }
            }


            // hang out until the gpu finishes ...
//            double wait = omp_get_wtime();
//            double accum = 0.0;
//            do {
//                if ( omp_get_wtime() - wait > 5.0 ) {
//                    accum += omp_get_wtime() - wait;
//                    wait = omp_get_wtime();
//                    outfile->Printf("gpu has taken an extra %6.2lf s\n",accum);
//                }
//            }while(!gpudone);

            //if (gpudone) helper_->GPUTiledDGEMM('n','n',o*v,o*v,o*v,1.0,tempv,o*v,tempt,o*v,0.0,integrals,o*v);
            //else F_DGEMM('n','n',o*v,o*v,o*v,1.0,tempv,o*v,tempt,o*v,0.0,integrals,o*v);
            gpuchunk = 0;
            odone    = 0;
            for (int i = 0; i < o; i++) {
                if (!gpudone) {
                    F_DGEMM('n','n',o*v,v,o*v,1.0,tempv,o*v,tempt+i*o*v*v,o*v,0.0,integrals+i*o*v*v,o*v);
                }else {
                    gpuchunk = o - i;
                    odone    = i;
                    break;
                }
            }
            if (gpudone && gpuchunk > 0) {
                helper_->GPUTiledDGEMM('n','n',o*v,gpuchunk*v,o*v,1.0,tempv,o*v,tempt+odone*o*v*v,o*v,0.0,integrals+odone*o*v*v,o*v);
            }

            if (gpudone) {
                omp_set_num_threads(ncputhreads);
                mkl_set_num_threads(ncputhreads);
                mkl_domain_set_num_threads(ncputhreads, MKL_DOMAIN_BLAS);
            }

            psio->open(PSIF_DCC_QSO,PSIO_OPEN_OLD);
            psio->read_entry(PSIF_DCC_QSO,"qvo",(char*)&tempv[0],nQ*o*v*sizeof(double));
            psio->close(PSIF_DCC_QSO,1);
            if (gpudone) helper_->GPUTiledDGEMM('n','t',o*v,o*v,nQ,2.0,Qov,o*v,tempv,o*v,1.0,integrals,o*v);
            else F_DGEMM('n','t',o*v,o*v,nQ,2.0,Qov,o*v,tempv,o*v,1.0,integrals,o*v);

            if (gpudone) {
                omp_set_num_threads(ncputhreads);
                mkl_set_num_threads(ncputhreads);
                mkl_domain_set_num_threads(ncputhreads, MKL_DOMAIN_BLAS);
            }

            if (gpudone) helper_->GPUTiledDGEMM('n','n',o*o,v*v,nQ,-1.0,Qoo,o*o,Qvv,nQ,0.0,tempv,o*o);
            else F_DGEMM('n','n',o*o,v*v,nQ,-1.0,Qoo,o*o,Qvv,nQ,0.0,tempv,o*o);

            if (gpudone) {
                omp_set_num_threads(ncputhreads);
                mkl_set_num_threads(ncputhreads);
                mkl_domain_set_num_threads(ncputhreads, MKL_DOMAIN_BLAS);
            }

            //F_DGEMM('n','t',o*o,v*v,nQ,-1.0,Qoo,o*o,Qvv,v*v,0.0,tempv,o*o);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int a = 0; a < v; a++) {
                for (int i = 0; i < o; i++) {
                    for (int k = 0; k < o; k++) {
                        for (int c = 0; c < v; c++) {
                            integrals[a*o*o*v+i*o*v+k*v+c] += tempv[a*o*o*v+c*o*o+k*o+i];
                        }
                    }
                }
            }
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int k = 0; k < o; k++) {
                for (int c = 0; c < v; c++) {
                    for (int b = 0; b < v; b++) {
                        for (int j = 0; j < o; j++) {
                            tempt[k*o*v*v+c*o*v+b*o+j] = 2.0 * tb[b*o*o*v+c*o*o+j*o+k] - tb[b*o*o*v+c*o*o+k*o+j];
                        }
                    }
                }
            }
            //if (gpudone) helper_->GPUTiledDGEMM('n','n',o*v,o*v,o*v,0.5,tempt,o*v,integrals,o*v,0.0,tempv,o*v);
            //else F_DGEMM('n','n',o*v,o*v,o*v,0.5,tempt,o*v,integrals,o*v,0.0,tempv,o*v);
            gpuchunk = 0;
            odone    = 0;
            for (int i = 0; i < o; i++) {
                if (!gpudone) {
                    F_DGEMM('n','n',o*v,v,o*v,0.5,tempt,o*v,integrals+i*o*v*v,o*v,0.0,tempv+i*o*v*v,o*v);
                }else {
                    gpuchunk = o - i;
                    odone    = i;
                    break;
                }
            }
            if (gpudone && gpuchunk > 0) {
                helper_->GPUTiledDGEMM('n','n',o*v,gpuchunk*v,o*v,0.5,tempt,o*v,integrals+odone*o*v*v,o*v,0.0,tempv+odone*o*v*v,o*v);
            }

            if (gpudone) {
                omp_set_num_threads(ncputhreads);
                mkl_set_num_threads(ncputhreads);
                mkl_domain_set_num_threads(ncputhreads, MKL_DOMAIN_BLAS);
            }

            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int a = 0; a < v; a++) {
                for (int b = 0; b < v; b++) {
                    for (int i = 0; i < o; i++) {
                        for (int j = 0; j < o; j++) {
                            tempt[a*o*o*v+b*o*o+i*o+j] = tempv[a*o*o*v+i*o*v+b*o+j];
                        }
                    }
                }
            }
            psio->open(PSIF_DCC_R2,PSIO_OPEN_OLD);
            psio->read_entry(PSIF_DCC_R2,"residual",(char*)&tempv[0],o*o*v*v*sizeof(double));
            C_DAXPY(o*o*v*v,1.0,tempv,1,tempt,1);
            psio->write_entry(PSIF_DCC_R2,"residual",(char*)&tempt[0],o*o*v*v*sizeof(double));
            psio->close(PSIF_DCC_R2,1);
            if (timer) {
                outfile->Printf("        D2 =  1/2 U(b,c,j,k) [ L(a,i,k,c) + 1/2 U(a,d,i,l) L(l,d,k,c) ] %6.2lf\n",omp_get_wtime()-start);
                start = omp_get_wtime();
            }

            if (gpudone) {
                omp_set_num_threads(ncputhreads);
                mkl_set_num_threads(ncputhreads);
                mkl_domain_set_num_threads(ncputhreads, MKL_DOMAIN_BLAS);
            }


            // E2 a: t(ac,ij) [ F(bc) - U(bd,kl) (ld|kc) ]
            C_DCOPY(o*o*v*v,tb,1,tempt,1);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int b = 0; b < v; b++) {
                for (int d = 0; d < v; d++) {
                    for (int k = 0; k < o; k++) {
                        C_DAXPY(o,-0.5,tb+b*o*o*v+d*o*o+k,o,tempt+b*o*o*v+d*o*o+k*o,1);
                    }
                }
            }
            if (gpudone) helper_->GPUTiledDGEMM('n','t',o*v,o*v,nQ,1.0,Qov,o*v,Qov,o*v,0.0,integrals,o*v);
            else F_DGEMM('n','t',o*v,o*v,nQ,1.0,Qov,o*v,Qov,o*v,0.0,integrals,o*v);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int c = 0; c < v; c++) {
                for (int d = 0; d < v; d++) {
                    for (int k = 0; k < o; k++) {
                        for (int l = 0; l < o; l++) {
                            tempv[c*o*o*v+d*o*o+k*o+l] = integrals[l*o*v*v+d*o*v+k*v+c];
                        }
                    }
                }
            }
            // overwriting Fab here, but it gets rebuilt every iteration anyway.
            if (gpudone) helper_->GPUTiledDGEMM('t','n',v,v,o*o*v,-2.0,tempv,o*o*v,tempt,o*o*v,1.0,Fab,v);
            else F_DGEMM('t','n',v,v,o*o*v,-2.0,tempv,o*o*v,tempt,o*o*v,1.0,Fab,v);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int c = 0; c < v; c++) {
                for (int a = 0; a < v; a++) {
                    for (int i = 0; i < o; i++) {
                        for (int j = 0; j < o; j++) {
                            tempt[c*o*o*v+a*o*o+i*o+j] = tb[a*o*o*v+c*o*o+i*o+j];
                        }
                    }
                }
            }
            if (gpudone) helper_->GPUTiledDGEMM('n','n',o*o*v,v,v,1.0,tempt,o*o*v,Fab,v,0.0,tempv,o*o*v);
            else F_DGEMM('n','n',o*o*v,v,v,1.0,tempt,o*o*v,Fab,v,0.0,tempv,o*o*v);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int a = 0; a < v; a++) {
                for (int b = 0; b < v; b++) {
                    for (int i = 0; i < o; i++) {
                        for (int j = 0; j < o; j++) {
                            tempt[a*o*o*v+b*o*o+i*o+j] = tempv[b*o*o*v+a*o*o+i*o+j];
                        }
                    }
                }
            }
            psio->open(PSIF_DCC_R2,PSIO_OPEN_OLD);
            psio->read_entry(PSIF_DCC_R2,"residual",(char*)&tempv[0],o*o*v*v*sizeof(double));
            C_DAXPY(o*o*v*v,1.0,tempv,1,tempt,1);
            psio->write_entry(PSIF_DCC_R2,"residual",(char*)&tempt[0],o*o*v*v*sizeof(double));
            psio->close(PSIF_DCC_R2,1);
            if (timer) {
                outfile->Printf("        E2 =      t(a,c,i,j) [ F(b,c) - U(b,d,k,l) (ld|kc) ]            %6.2lf\n",omp_get_wtime()-start);
                start = omp_get_wtime();
            }

            // E2 b: -t(a,b,i,k) [ F(kj) - U(c,d,l,j) (kd|lc) ]
            // note that (kd|lc) should still be in integrals buffer
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int j = 0; j < o; j++) {
                for (int d = 0; d < v; d++) {
                    for (int l = 0; l < o; l++) {
                        for (int c = 0; c < v; c++) {
                            tempt[j*o*v*v+d*o*v+l*v+c] = (2.0 * tb[c*o*o*v+d*o*o+l*o+j] - tb[c*o*o*v+d*o*o+j*o+l] );
                        }
                    }
                }
            }
            // overwriting Fij here, but it gets rebuilt every iteration anyway.
            if (gpudone) helper_->GPUTiledDGEMM('t','n',o,o,o*v*v,1.0,tempt,o*v*v,integrals,o*v*v,1.0,Fij,o);
            else F_DGEMM('t','n',o,o,o*v*v,1.0,tempt,o*v*v,integrals,o*v*v,1.0,Fij,o);

            psio->open(PSIF_DCC_R2,PSIO_OPEN_OLD);
            psio->read_entry(PSIF_DCC_R2,"residual",(char*)&tempt[0],o*o*v*v*sizeof(double));
            //if (gpudone) helper_->GPUTiledDGEMM('n','n',o,o*v*v,o,-1.0,Fij,o,tb,o,1.0,tempt,o);
            //else F_DGEMM('n','n',o,o*v*v,o,-1.0,Fij,o,tb,o,1.0,tempt,o);
            F_DGEMM('n','n',o,o*v*v,o,-1.0,Fij,o,tb,o,1.0,tempt,o);

            // R2 = R2 + P(ia,jb) R2
            C_DCOPY(o*o*v*v,tempt,1,integrals,1);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int a = 0; a < v; a++) {
                for (int b = 0; b < v; b++) {
                    for (int i = 0; i < o; i++) {
                        for (int j = 0; j < o; j++) {
                            integrals[a*o*o*v+b*o*o+i*o+j] += tempt[b*o*o*v+a*o*o+j*o+i];
                        }
                    }
                }
            }
            psio->write_entry(PSIF_DCC_R2,"residual",(char*)&integrals[0],o*o*v*v*sizeof(double));
            psio->close(PSIF_DCC_R2,1);
            if (timer) {
                outfile->Printf("                - t(a,b,i,k) [ F(k,j) - U(c,d,l,j) (kd|lc) ]            %6.2lf\n",omp_get_wtime()-start);
                start = omp_get_wtime();
            }

            // B2 = t(ab,kl) [ (ki|lj) + t(cd,ij) (kc|ld) ]
            if (gpudone) helper_->GPUTiledDGEMM('n','t',o*v,o*v,nQ,1.0,Qov,o*v,Qov,o*v,0.0,integrals,o*v);
            else F_DGEMM('n','t',o*v,o*v,nQ,1.0,Qov,o*v,Qov,o*v,0.0,integrals,o*v);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int k = 0; k < o; k++) {
                for (int l = 0; l < o; l++) {
                    for (int c = 0; c < v; c++) {
                        for (int d = 0; d < v; d++) {
                            tempv[k*o*v*v+l*v*v+c*v+d] = integrals[k*v*v*o+c*o*v+l*v+d];
                        }
                    }
                }
            }
            if (gpudone) helper_->GPUTiledDGEMM('n','t',o*o,o*o,nQ,1.0,Qoo,o*o,Qoo,o*o,0.0,integrals,o*o);
            else F_DGEMM('n','t',o*o,o*o,nQ,1.0,Qoo,o*o,Qoo,o*o,0.0,integrals,o*o);
            #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
            for (int k = 0; k < o; k++) {
                for (int i = 0; i < o; i++) {
                    for (int l = 0; l < o; l++) {
                        for (int j = 0; j < o; j++) {
                            tempt[k*o*o*o+l*o*o+i*o+j] = integrals[k*o*o*o+i*o*o+l*o+j];
                        }
                    }
                }
            }
            if (gpudone) helper_->GPUTiledDGEMM('n','n',o*o,o*o,v*v,1.0,tb,o*o,tempv,v*v,1.0,tempt,o*o);
            else F_DGEMM('n','n',o*o,o*o,v*v,1.0,tb,o*o,tempv,v*v,1.0,tempt,o*o);
            if (gpudone) helper_->GPUTiledDGEMM('n','n',o*o,v*v,o*o,1.0,tempt,o*o,tb,o*o,0.0,integrals,o*o);
            else F_DGEMM('n','n',o*o,v*v,o*o,1.0,tempt,o*o,tb,o*o,0.0,integrals,o*o);

            psio->open(PSIF_DCC_R2,PSIO_OPEN_OLD);
            psio->read_entry(PSIF_DCC_R2,"residual",(char*)&tempt[0],o*o*v*v*sizeof(double));
            C_DAXPY(o*o*v*v,1.0,tempt,1,integrals,1);
            psio->write_entry(PSIF_DCC_R2,"residual",(char*)&integrals[0],o*o*v*v*sizeof(double));
            psio->close(PSIF_DCC_R2,1);

            if (timer) {
                outfile->Printf("        B2 =      t(a,b,k,l) [ (ki|lj) + t(c,d,i,j) (kc|ld) ]           %6.2lf\n",omp_get_wtime()-start);
                start = omp_get_wtime();
            }

            cpudone = true;
//*/
//AED
        }
        //////// end cpu section! ////////

}

void GPUDFCoupledCluster::CCResidual(){
    bool timer = options_.get_bool("CC_TIMINGS");
    long int v = nvirt;

    // test new transposed storage of Qvv

    // qvv transpose
    #pragma omp parallel for schedule (static)
    for (int q = 0; q < nQ; q++) {
        C_DCOPY(v*v,Qvv+q*v*v,1,integrals+q,nQ);
    }
    C_DCOPY(nQ*v*v,integrals,1,Qvv,1);


    int n = 2 < omp_get_max_threads() ? 2 : omp_get_max_threads();


    pthread_attr_t pthread_custom_attr;
    pthread_t * threads = (pthread_t *)malloc(n*sizeof(*threads));
    pthread_attr_init(&pthread_custom_attr);
    
    mega * m = (mega *)malloc(sizeof(mega)*n);

    gpudone     = false;
    cpudone     = false;

    double start = omp_get_wtime();
    for (int i = 0; i < n; i++) {
        m[i].id = i;
        m[i].cc = (&(*this));
        pthread_create(&threads[i], &pthread_custom_attr, doit, (void*)(m+i));
    }

    // Synchronize the completion of each thread.
    for (int i = 0; i < n; i++) {
    	pthread_join(threads[i],NULL);
    }
    free(threads);

    // it is possible the gpu didn't finish its work.  check and finish with 
    // the CPU and the GPU together
    if (cpudone && !gpudone) {
        //outfile->Printf("cpu finished first! %5i %5i\n",v,last_a);fflush(stdout);//exit(0);
        FinishVabcd1();
        if (timer) {
            outfile->Printf("        A2 =      t(c,d,i,j) (ac|bd)                                    %6.2lf\n",omp_get_wtime()-start);
        }
    }

    // use results of contraction of (ac|bd) and t2
    useVabcd1();

}

// t1-transformed 3-index fock matrix (using 3-index integrals from SCF)
void GPUDFCoupledCluster::T1Fock(){
    long int o = ndoccact;
    long int v = nvirt;
    long int full = o+v+nfzc+nfzv;

    // Ca_L = C(1-t1^T)
    // Ca_R = C(1+t1)
    double * Catemp = (double*)malloc(nso*full*sizeof(double));
    C_DCOPY(nso*full,&Ca[0][0],1,Ca_L,1);
    C_DCOPY(nso*full,&Ca[0][0],1,Ca_R,1);
    C_DCOPY(nso*full,&Ca[0][0],1,Catemp,1);
   

    #pragma omp parallel for schedule (static)
    for (int mu = 0; mu < nso; mu++) {
        for (int a = 0; a < v; a++) {
            double dum = 0.0;
            for (int i = 0; i < o; i++) {
                dum += Catemp[mu*full+i+nfzc] * t1[a*o+i];
            }
            Ca_L[mu*full + a + ndocc] -= dum;
        }
    }
    #pragma omp parallel for schedule (static)
    for (int mu = 0; mu < nso; mu++) {
        for (int i = 0; i < o; i++) {
            double dum = 0.0;
            for (int a = 0; a < v; a++) {
                dum += Catemp[mu*full+a+ndocc] * t1[a*o+i];
            }
            Ca_R[mu*full + i + nfzc] += dum;
        }
    }
    free(Catemp);

    // (Q|rs)
    std::shared_ptr<PSIO> psio(new PSIO());
    psio->open(PSIF_DCC_QSO,PSIO_OPEN_OLD);
    psio_address addr1  = PSIO_ZERO;
    psio_address addr2  = PSIO_ZERO;

    long int nrows = 1;
    long int rowsize = nQ_scf;
    while ( rowsize*nso*nso > o*o*v*v ) {
        nrows++;
        rowsize = nQ_scf / nrows;
        if (nrows * rowsize < nQ_scf) rowsize++;
        if (rowsize == 1) break;
    }
    long int lastrowsize = nQ_scf - (nrows - 1L) * rowsize;
    long int * rowdims = new long int [nrows];
    for (int i = 0; i < nrows-1; i++) rowdims[i] = rowsize;
    rowdims[nrows-1] = lastrowsize;
    for (int row = 0; row < nrows; row++) {
        psio->read(PSIF_DCC_QSO,"Qso SCF",(char*)&integrals[0],rowdims[row]*nso*nso*sizeof(double),addr1,&addr1);
        F_DGEMM('n','n',full,nso*rowdims[row],nso,1.0,Ca_L,full,integrals,nso,0.0,tempv,full);
        for (int q = 0; q < rowdims[row]; q++) {
            for (int mu = 0; mu < nso; mu++) {
                C_DCOPY(full,tempv+q*nso*full+mu*full,1,integrals+q*nso*full+mu,nso);
            }
        }
        F_DGEMM('n','n',full,full*rowdims[row],nso,1.0,Ca_R,full,integrals,nso,0.0,tempv,full);
        // full Qmo
        psio->write(PSIF_DCC_QSO,"Qmo SCF",(char*)&tempv[0],rowdims[row]*full*full*sizeof(double),addr2,&addr2);
    }
    delete rowdims;

    // build Fock matrix

    memset((void*)Fij,'\0',o*o*sizeof(double));
    memset((void*)Fia,'\0',o*v*sizeof(double));
    memset((void*)Fai,'\0',o*v*sizeof(double));
    memset((void*)Fab,'\0',v*v*sizeof(double));

    // transform H
    double ** hp = H->pointer();
    double * h = (double*)malloc(nmo*nmo*sizeof(double));
    for (int mu = 0; mu < nso; mu++) {
        for (int p = 0; p < nmo; p++) {
            double dum = 0.0;
            for (int nu = 0; nu < nso; nu++) {
                dum += Ca_L[nu*full + p + nfzc] * hp[nu][mu];
            }
            integrals[p*nso+mu] = dum;
        }
    }
    for (int p = 0; p < nmo; p++) {
        for (int q = 0; q < nmo; q++) {
            double dum = 0.0;
            for (int nu = 0; nu < nso; nu++) {
                dum += Ca_R[nu*full+q+nfzc] * integrals[p*nso+nu];
            }
            h[p*nmo+q] = dum;
        }
    }

    double * temp3 = (double*)malloc(full*full*sizeof(double));

    memset((void*)temp3,'\0',full*full*sizeof(double));
    psio_address addr = PSIO_ZERO;

    nrows = 1;
    rowsize = nQ_scf;
    while ( rowsize*full*full > o*o*v*v ) {
        nrows++;
        rowsize = nQ_scf / nrows;
        if (nrows * rowsize < nQ_scf) rowsize++;
        if (rowsize == 1) break;
    }
    lastrowsize = nQ_scf - (nrows - 1L) * rowsize;
    rowdims = new long int [nrows];
    for (int i = 0; i < nrows-1; i++) rowdims[i] = rowsize;
    rowdims[nrows-1] = lastrowsize;
    for (int row = 0; row < nrows; row++) {
        psio->read(PSIF_DCC_QSO,"Qmo SCF",(char*)&integrals[0],rowdims[row]*full*full*sizeof(double),addr,&addr);
        for (int q = 0; q < rowdims[row]; q++) {
            // sum k (q|rk) (q|ks)
            F_DGEMM('n','n',full,full,ndocc,-1.0,integrals+q*full*full,full,integrals+q*full*full,full,1.0,temp3,full);

            // sum k (q|kk) (q|rs)
            double dum = 0.0;
            for (int k = 0; k < ndocc; k++) {
                dum += integrals[q*full*full+k*full + k];
            }
            C_DAXPY(full*full,2.0 * dum,integrals+q*full*full,1,temp3,1);
        }
    }
    delete rowdims;
    psio->close(PSIF_DCC_QSO,1);

    // Fij
    for (int i = 0; i < o; i++) {
        for (int j = 0; j < o; j++) {
            Fij[i*o+j] = h[i*nmo+j] + temp3[(i+nfzc)*full+(j+nfzc)];
        }
    }

    // Fia
    for (int i = 0; i < o; i++) {
        for (int a = 0; a < v; a++) {
            Fia[i*v+a] = h[i*nmo+a+o] + temp3[(i+nfzc)*full+(a+ndocc)];
        }
    }

    // Fai
    for (int a = 0; a < v; a++) {
        for (int i = 0; i < o; i++) {
            Fai[a*o+i] = h[(a+o)*nmo+i] + temp3[(a+ndocc)*full+(i+nfzc)];
        }
    }

    // Fab
    for (int a = 0; a < v; a++) {
        for (int b = 0; b < v; b++) {
            Fab[a*v+b] = h[(a+o)*nmo+b+o] + temp3[(a+ndocc)*full+(b+ndocc)];
        }
    }

    // replace eps
    for (int i = 0; i < o; i++) {
        eps[i] = Fij[i*o+i];
    }
    for (int a = 0; a < v; a++) {
        eps[a+o] = Fab[a*v+a];
    }
    free(h);
    free(temp3);

}

// t1-transformed 3-index integrals
void GPUDFCoupledCluster::T1Integrals(){
    long int o = ndoccact;
    long int v = nvirt;
    long int full = o+v+nfzc+nfzv;

    // Ca_L = C(1-t1^T)
    // Ca_R = C(1+t1)
    double * Catemp = (double*)malloc(nso*full*sizeof(double));
    C_DCOPY(nso*full,&Ca[0][0],1,Ca_L,1);
    C_DCOPY(nso*full,&Ca[0][0],1,Ca_R,1);
    C_DCOPY(nso*full,&Ca[0][0],1,Catemp,1);

    #pragma omp parallel for schedule (static)
    for (int mu = 0; mu < nso; mu++) {
        for (int a = 0; a < v; a++) {
            double dum = 0.0;
            for (int i = 0; i < o; i++) {
                dum += Catemp[mu*full+i+nfzc] * t1[a*o+i];
            }
            Ca_L[mu*full + a + ndocc] -= dum;
        }
    }
    #pragma omp parallel for schedule (static)
    for (int mu = 0; mu < nso; mu++) {
        for (int i = 0; i < o; i++) {
            double dum = 0.0;
            for (int a = 0; a < v; a++) {
                dum += Catemp[mu*full+a+ndocc] * t1[a*o+i];
            }
            Ca_R[mu*full + i + nfzc] += dum;
        }
    }
    free(Catemp);

    // (Q|rs)
    std::shared_ptr<PSIO> psio(new PSIO());
    psio->open(PSIF_DCC_QSO,PSIO_OPEN_OLD);
    psio_address addr1  = PSIO_ZERO;
    psio_address addrvo = PSIO_ZERO;
    long int nrows = 1;
    long int rowsize = nQ;
    while ( rowsize*nso*nso > o*o*v*v ) {
        nrows++;
        rowsize = nQ / nrows;
        if (nrows * rowsize < nQ) rowsize++;
        if ( rowsize == 1 ) break;
    }
    long int lastrowsize = nQ - (nrows - 1L) * rowsize;
    long int * rowdims = new long int [nrows];
    for (int i = 0; i < nrows-1; i++) rowdims[i] = rowsize;
    rowdims[nrows-1] = lastrowsize;
    for (int row = 0; row < nrows; row++) {
        psio->read(PSIF_DCC_QSO,"Qso CC",(char*)&integrals[0],rowdims[row]*nso*nso*sizeof(double),addr1,&addr1);
        //helper_->GPUTiledDGEMM('n','n',full,nso*rowdims[row],nso,1.0,Ca_L,full,integrals,nso,0.0,tempv,full);
        F_DGEMM('n','n',full,nso*rowdims[row],nso,1.0,Ca_L,full,integrals,nso,0.0,tempv,full);
        for (int q = 0; q < rowdims[row]; q++) {
            for (int mu = 0; mu < nso; mu++) {
                C_DCOPY(full,tempv+q*nso*full+mu*full,1,integrals+q*nso*full+mu,full);
            }
        }
        //helper_->GPUTiledDGEMM('n','n',full,full*rowdims[row],nso,1.0,Ca_R,full,integrals,nso,0.0,tempv,full);
        F_DGEMM('n','n',full,full*rowdims[row],nso,1.0,Ca_R,full,integrals,nso,0.0,tempv,full);

        // Qoo
        #pragma omp parallel for schedule (static)
        for (int q = 0; q < rowdims[row]; q++) {
            for (int i = 0; i < o; i++) {
                for (int j = 0; j < o; j++) {
                    Qoo[(q+rowdims[0]*row)*o*o+i*o+j] = tempv[q*full*full+(i+nfzc)*full+(j+nfzc)];
                }
            }
        }
        // Qov
        #pragma omp parallel for schedule (static)
        for (int q = 0; q < rowdims[row]; q++) {
            for (int i = 0; i < o; i++) {
                for (int a = 0; a < v; a++) {
                    Qov[(q+rowdims[0]*row)*o*v+i*v+a] = tempv[q*full*full+(i+nfzc)*full+(a+ndocc)];
                }
            }
        }
        // Qvo
        #pragma omp parallel for schedule (static)
        for (int q = 0; q < rowdims[row]; q++) {
            for (int a = 0; a < v; a++) {
                for (int i = 0; i < o; i++) {
                    integrals[q*o*v+a*o+i] = tempv[q*full*full+(a+ndocc)*full+(i+nfzc)];
                }
            }
        }
        psio->write(PSIF_DCC_QSO,"qvo",(char*)&integrals[0],rowdims[row]*o*v*sizeof(double),addrvo,&addrvo);
        // Qvv
        #pragma omp parallel for schedule (static)
        for (int q = 0; q < rowdims[row]; q++) {
            for (int a = 0; a < v; a++) {
                for (int b = 0; b < v; b++) {
                    Qvv[(q+rowdims[0]*row)*v*v+a*v+b] = tempv[q*full*full+(a+ndocc)*full+(b+ndocc)];
                }
            }
        }
    }
    delete rowdims;
    psio->close(PSIF_DCC_QSO,1);


    // check mp2 energy
    /*double * tints = (double*)malloc(o*o*v*v*sizeof(double));
    memset((void*)tints,'\0',o*o*v*v*sizeof(double));
    F_DGEMM('n','t',o*v,o*v,nQ,1.0,Qov,o*v,Qov,o*v,0.0,tints,o*v);
    double e2 = 0.0;
    for (int i = 0; i < o; i++) {
        for (int j = 0; j < o; j++) {
            for (int a = 0; a < v; a++) {
                for (int b = 0; b < v; b++) {
                    double dijab = (eps[i] + eps[j] - eps[a+o] - eps[b+o]);
                    long int iajb = i*v*v*o + a*o*v + j*v + b;
                    long int jaib = j*v*v*o + a*v*o + i*v + b;
                    e2 += (2.0 * tints[iajb] - tints[jaib]) * tints[iajb] / dijab;
                }
            }
        }
    }
    printf("mp2 energy %20.12lf\n",e2);
    //exit(0);*/



    // check ccsd energy
  /*  F_DGEMM('n','t',o*v,o*v,nQ,1.0,Qov,o*v,Qov,o*v,0.0,integrals,o*v);

    if (t2_on_disk){
        std::shared_ptr<PSIO> psio (new PSIO());
        psio->open(PSIF_DCC_T2,PSIO_OPEN_OLD);
        psio->read_entry(PSIF_DCC_T2,"t2",(char*)&tempv[0],o*o*v*v*sizeof(double));
        psio->close(PSIF_DCC_T2,1);
        tb = tempv;
    }
    double energy = 0.0;
    double mp2energy = 0.0;
    for (long int a = 0; a < v; a++){
        for (long int b = 0; b < v; b++){
            for (long int i = 0; i < o; i++){
                for (long int j = 0; j < o; j++){
                    double dijab = (eps[i] + eps[j] - eps[a+o] - eps[b+o]);
                    long int ijab = a*v*o*o + b*o*o + i*o + j;
                    long int iajb = i*v*v*o + a*v*o + j*v + b;
                    long int jaib = j*v*v*o + a*v*o + i*v + b;
                    energy += (2.0*integrals[iajb]-integrals[jaib])*tb[ijab];
                    mp2energy += (2.0*integrals[iajb]-integrals[jaib])*integrals[iajb]/dijab;
                }
            }
        }
    }
    printf("ccsd energy %20.12lf\n",energy);
    printf("mp2 energy  %20.12lf\n",mp2energy);
*/
}

double GPUDFCoupledCluster::compute_energy() {
  PsiReturnType status = Success;

  //WriteBanner();
  AllocateMemory();
  status = CCSDIterations();

  // free some memory!
  free(Fij);
  free(Fab);
  free(Abij);
  free(Sbij);
  free(integrals);
  free(w1);
  free(I1);
  free(I1p);
  free(diisvec);
  free(tempt);
  free(tempv);

  // tstart in fnocc
  tstop();

  // mp2 energy
  Process::environment.globals["MP2 CORRELATION ENERGY"] = emp2;
  Process::environment.globals["MP2 TOTAL ENERGY"] = emp2 + escf;
  Process::environment.globals["MP2 OPPOSITE-SPIN CORRELATION ENERGY"] = emp2_os;
  Process::environment.globals["MP2 SAME-SPIN CORRELATION ENERGY"] = emp2_ss;

  // ccsd energy
  Process::environment.globals["CCSD CORRELATION ENERGY"] = eccsd;
  Process::environment.globals["CCSD OPPOSITE-SPIN CORRELATION ENERGY"] = eccsd_os;
  Process::environment.globals["CCSD SAME-SPIN CORRELATION ENERGY"] = eccsd_ss;
  Process::environment.globals["CCSD TOTAL ENERGY"] = eccsd + escf;
  Process::environment.globals["CURRENT ENERGY"] = eccsd + escf;

  if (options_.get_bool("COMPUTE_TRIPLES")){
      long int o = ndoccact;
      long int v = nvirt;

      if (!isLowMemory ) {
          // write (ov|vv) integrals, formerly E2abci, for (t)
          double *tempq = (double*)malloc(v*nQ*sizeof(double));
          // the buffer integrals was at least 2v^3, so these should definitely fit.
          double *Z     = (double*)malloc(v*v*v*sizeof(double));
          double *Z2    = (double*)malloc(v*v*v*sizeof(double));
          std::shared_ptr<PSIO> psio(new PSIO());
          psio->open(PSIF_DCC_ABCI,PSIO_OPEN_NEW);
          psio_address addr2 = PSIO_ZERO;
          for (long int i=0; i<o; i++){
              #pragma omp parallel for schedule (static)
              for (long int q=0; q<nQ; q++){
                  for (long int b=0; b<v; b++){
                      tempq[q*v+b] = Qov[q*o*v+i*v+b];
                  }
              }     
              helper_->GPUTiledDGEMM('n','t',v,v*v,nQ,1.0,tempq,v,Qvv,v*v,0.0,&Z[0],v);
              #pragma omp parallel for schedule (static)
              for (long int a=0; a<v; a++){
                  for (long int b=0; b<v; b++){
                      for (long int c=0; c<v; c++){
                          Z2[a*v*v+b*v+c] = Z[a*v*v+c*v+b];
                      }
                  }
              }
              psio->write(PSIF_DCC_ABCI,"E2abci",(char*)&Z2[0],v*v*v*sizeof(double),addr2,&addr2);
          }
          psio->close(PSIF_DCC_ABCI,1);
          free(tempq);
          free(Z);
          free(Z2);
      } else {
          psio_address addr = PSIO_ZERO;
          double * temp1 = (double*)malloc(( nQ*v > o*v*v ? nQ*v : o*v*v)*sizeof(double));
          double * temp2 = (double*)malloc(o*v*v*sizeof(double));
          std::shared_ptr<PSIO> psio(new PSIO());
          psio->open(PSIF_DCC_ABCI4,PSIO_OPEN_NEW);
          for (long int a = 0; a < v; a++) {
              #pragma omp parallel for schedule (static)
              for (long int q = 0; q < nQ; q++) {
                  for (long int c = 0; c < v; c++) {
                      temp1[q*v+c] = Qvv[q*v*v+a*v+c];
                  }
              }
              helper_->GPUTiledDGEMM('n','t',o*v,v,nQ,1.0,Qov,o*v,temp1,v,0.0,temp2,o*v);
              #pragma omp parallel for schedule (static)
              for (long int b = 0; b < v; b++) {
                  for (long int i = 0; i < o; i++) {
                      for (long int c = 0; c < v; c++) {
                          temp1[b*o*v+i*v+c] = temp2[c*o*v+i*v+b];
                      }
                  }
              }
              psio->write(PSIF_DCC_ABCI4,"E2abci4",(char*)&temp1[0],o*v*v*sizeof(double),addr,&addr);
          }
          psio->close(PSIF_DCC_ABCI4,1);
          free(temp1);
          free(temp2);
      }
      hipHostFree(Qvv);//free(Qvv);
      double * temp1 = (double*)malloc(o*o*v*v*sizeof(double));
      double * temp2 = (double*)malloc(o*o*v*v*sizeof(double));

      // write (oo|ov) integrals, formerly E2ijak, for (t)
      helper_->GPUTiledDGEMM('n','t',o*o,o*v,nQ,1.0,Qoo,o*o,Qov,o*v,0.0,temp1,o*o);
      for (int i=0; i<o; i++){
          for (int j=0; j<o; j++){
              for (int k=0; k<o; k++){
                  for (int a=0; a<v; a++){
                      temp2[j*o*o*v+i*o*v+k*v+a] = temp1[i*o*o*v+a*o*o+j*o+k];
                  }
              }
          }
      }
      std::shared_ptr<PSIO> psio(new PSIO());
      psio->open(PSIF_DCC_IJAK,PSIO_OPEN_NEW);
      psio->write_entry(PSIF_DCC_IJAK,"E2ijak",(char*)&temp2[0],o*o*o*v*sizeof(double));
      psio->close(PSIF_DCC_IJAK,1);

      // df (ov|ov) integrals, formerly E2klcd
      helper_->GPUTiledDGEMM('n','t',o*v,o*v,nQ,1.0,Qov,o*v,Qov,o*v,0.0,temp1,o*v);
      psio->open(PSIF_DCC_IAJB,PSIO_OPEN_NEW);
      psio->write_entry(PSIF_DCC_IAJB,"E2iajb",(char*)&temp1[0],o*o*v*v*sizeof(double));
      psio->close(PSIF_DCC_IAJB,1);

      free(Qov);
      free(Qoo);
      free(temp1);
      free(temp2);

      // triples
      tstart();

      ccmethod = 0;
      if (isLowMemory)                           status = lowmemory_triples();
      else                                       status = triples();

      if (status == Failure){
         throw PsiException(
            "Whoops, the (T) correction died.",__FILE__,__LINE__);
      }
      tstop();

      // ccsd(t) energy
      Process::environment.globals["(T) CORRECTION ENERGY"] = et;
      Process::environment.globals["CCSD(T) CORRELATION ENERGY"] = eccsd + et;
      Process::environment.globals["CCSD(T) TOTAL ENERGY"] = eccsd + et + escf;
      Process::environment.globals["CURRENT ENERGY"] = eccsd + et + escf;
  }else {
      free(Qoo);
      free(Qov);
      free(Qvv);
  }

  // free remaining memory
  free(Fia);
  free(Fai);
  free(t1);
  free(tb);

  return Process::environment.globals["CURRENT ENERGY"];
}

void GPUDFCoupledCluster::UpdateT2(){
    long int v = nvirt;
    long int o = ndoccact;
    long int rs = nmo;

    std::shared_ptr<PSIO> psio(new PSIO());

    // df (ai|bj)
    psio->open(PSIF_DCC_QSO,PSIO_OPEN_OLD);
    psio->read_entry(PSIF_DCC_QSO,"qvo",(char*)&tempv[0],nQ*o*v*sizeof(double));
    psio->close(PSIF_DCC_QSO,1);
    helper_->GPUTiledDGEMM('n','t',o*v,o*v,nQ,1.0,tempv,o*v,tempv,o*v,0.0,integrals,o*v);

    // residual
    psio->open(PSIF_DCC_R2,PSIO_OPEN_OLD);
    psio->read_entry(PSIF_DCC_R2,"residual",(char*)&tempv[0],o*o*v*v*sizeof(double));
    psio->close(PSIF_DCC_R2,1);

    #pragma omp parallel for schedule (static)
    for (long int a=o; a<rs; a++){
        double da = eps[a];
        for (long int b=o; b<rs; b++){
            double dab = da + eps[b];
            for (long int i=0; i<o; i++){
                double dabi = dab - eps[i];
                for (long int j=0; j<o; j++){

                    long int iajb = (a-o)*v*o*o+i*v*o+(b-o)*o+j;
                    long int ijab = (a-o)*v*o*o+(b-o)*o*o+i*o+j;

                    double dijab = dabi-eps[j];
                    double tnew  = - (integrals[iajb] + tempv[ijab])/dijab;
                    //double tnew  = - (integrals[iajb])/dijab;
                    //tempt[ijab]  = tnew;
                    tempv[ijab]  = tnew;
                }
            }
        }
    }
    // error vector is just dt
    //C_DCOPY(o*o*v*v,tempt,1,tempv,1);

    if (t2_on_disk){
        psio->open(PSIF_DCC_T2,PSIO_OPEN_OLD);
        psio->read_entry(PSIF_DCC_T2,"t2",(char*)&integrals[0],o*o*v*v*sizeof(double));
        C_DAXPY(o*o*v*v,1.0,tempv,1,integrals,1);
        psio->write_entry(PSIF_DCC_T2,"t2",(char*)&integrals[0],o*o*v*v*sizeof(double));
        psio->close(PSIF_DCC_T2,1);
    }else {
        C_DAXPY(o*o*v*v,1.0,tempv,1,tb,1);
    }
}



}}
