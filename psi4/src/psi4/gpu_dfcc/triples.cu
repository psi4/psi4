/*
 *@BEGIN LICENSE
 *
 * GPU-accelerated density-fitted coupled-cluster, a plugin to:
 *
 * PSI4: an ab initio quantum chemistry software package
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 *
 *@END LICENSE
 */

#include"ccsd.h"
#include"blas.h"
#include"psi4/libmints/wavefunction.h"
#include"psi4/libqt/qt.h"
#include"psi4/libpsi4util/process.h"
#include"omp.h"


using namespace psi;

namespace psi{namespace fnocc{

PsiReturnType GPUDFCoupledCluster::triples(){
  char*name = new char[10];
  char*space = new char[10];
  double fac;
  if (ccmethod == 0) {
     sprintf(name,"CCSD");
     sprintf(space," ");
     fac = 1.0;
  }else if (ccmethod == 1) {
     sprintf(name,"QCISD");
     sprintf(space,"  ");
     fac = 2.0;
  }else{
     sprintf(name,"MP4");
     //sprintf(space,"");
     fac = 0.0;
  }

  outfile->Printf("\n");
  outfile->Printf( "        *******************************************************\n");
  outfile->Printf( "        *                                                     *\n");
  outfile->Printf( "        *                  %8s(T)                        *\n",name);
  outfile->Printf( "        *                                                     *\n");
  outfile->Printf( "        *******************************************************\n");
  outfile->Printf("\n");
  //fflush(outfile);

  int o = ndoccact;
  int v = nvirt_no;

  double *F  = eps;
  double *E2ijak,**E2abci;
  E2ijak = (double*)malloc(o*o*o*v*sizeof(double));
  int nthreads = 1;
  #ifdef _OPENMP
      nthreads = omp_get_max_threads();
  #endif

  long int memory = Process::environment.get_memory();
  if (options_["MEMORY"].has_changed()){
     memory  = options_.get_int("MEMORY");
     memory *= (long int)1024*1024;
  }
  memory -= 8L*(2L*o*o*v*v+o*o*o*v+o*v+3L*nthreads*v*v*v);

  outfile->Printf("        num_threads =             %9i\n",nthreads);
  outfile->Printf("        available memory =     %9.2lf mb\n",memory/1024./1024.);
  outfile->Printf("        memory requirements =  %9.2lf mb\n",
           8.*(2.*o*o*v*v+1.*o*o*o*v+(3.*nthreads)*v*v*v+1.*o*v)/1024./1024.);
  outfile->Printf("\n");
  //fflush(outfile);

  int nijk = 0;
  for (int i=0; i<o; i++){
      for (int j=0; j<=i; j++){
          for (int k=0; k<=j; k++){
              nijk++;
          }
      }
  }
  int**ijk = (int**)malloc(nijk*sizeof(int*));
  nijk = 0;
  for (int i=0; i<o; i++){
      for (int j=0; j<=i; j++){
          for (int k=0; k<=j; k++){
              ijk[nijk] = (int*)malloc(3*sizeof(int));
              ijk[nijk][0] = i;
              ijk[nijk][1] = j;
              ijk[nijk][2] = k;
              nijk++;
          }
      }
  }
  outfile->Printf("        Number of ijk combinations: %i\n",nijk);
  outfile->Printf("\n");
  //fflush(outfile);
  
  E2abci = (double**)malloc(nthreads*sizeof(double*));
  // some v^3 intermediates
  double **Z  = (double**)malloc(nthreads*sizeof(double*));
  double **Z2 = (double**)malloc(nthreads*sizeof(double*));

  for (int i=0; i<nthreads; i++){
      E2abci[i] = (double*)malloc(v*v*v*sizeof(double));
      Z[i]      = (double*)malloc(v*v*v*sizeof(double));
      Z2[i]     = (double*)malloc(v*v*v*sizeof(double));
  }

  std::shared_ptr<PSIO> psio(new PSIO());

  psio->open(PSIF_DCC_IJAK,PSIO_OPEN_OLD);
  psio->read_entry(PSIF_DCC_IJAK,"E2ijak",(char*)&E2ijak[0],o*o*o*v*sizeof(double));
  psio->close(PSIF_DCC_IJAK,1);

  double *tempt = (double*)malloc(o*o*v*v*sizeof(double));

  // first-order amplitudes for mp4
  if (ccmethod == 2) {
     psio->open(PSIF_DCC_T2,PSIO_OPEN_OLD);
     psio->read_entry(PSIF_DCC_T2,"first",(char*)&tb[0],o*o*v*v*sizeof(double));
     psio->close(PSIF_DCC_T2,1);
  }

  for (int a=0; a<v*v; a++){
      C_DCOPY(o*o,tb+a*o*o,1,tempt+a,v*v);
  }

  // might as well use t2's memory
  double*E2klcd = tb;
  psio->open(PSIF_DCC_IAJB,PSIO_OPEN_OLD);
  psio->read_entry(PSIF_DCC_IAJB,"E2iajb", (char*)&E2klcd[0],o*o*v*v*sizeof(double));
  psio->close(PSIF_DCC_IAJB,1);

  double *etrip = (double*)malloc(nthreads*sizeof(double));
  for (int i=0; i<nthreads; i++) etrip[i] = 0.0;
  outfile->Printf("        Computing (T) correction...\n");
  outfile->Printf("\n");
  outfile->Printf("        %% complete  total time\n");
  //fflush(outfile);

  time_t stop,start = time(NULL);
  int pct10,pct20,pct30,pct40,pct50,pct60,pct70,pct80,pct90;
  pct10=pct20=pct30=pct40=pct50=pct60=pct70=pct80=pct90=0;
  int pct01 = 0;
  int pct02 = 0;
  int pct03 = 0;
  int pct04 = 0;
  int pct05 = 0;

  /**
    *  if there is enough memory to explicitly thread, do so
    */
  #pragma omp parallel for schedule (dynamic) num_threads(nthreads)
  for (int ind=0; ind<nijk; ind++){
      int i = ijk[ind][0];
      int j = ijk[ind][1];
      int k = ijk[ind][2];

      int thread = 0;
      #ifdef _OPENMP
          thread = omp_get_thread_num();
      #endif

      std::shared_ptr<PSIO> mypsio(new PSIO());
      mypsio->open(PSIF_DCC_ABCI,PSIO_OPEN_OLD);

      psio_address addr = psio_get_address(PSIO_ZERO,(long int)k*v*v*v*sizeof(double));
      mypsio->read(PSIF_DCC_ABCI,"E2abci",(char*)&E2abci[thread][0],v*v*v*sizeof(double),addr,&addr);
      helper_->GPUTiledDGEMM_NoThread('t','t',v*v,v,v,1.0,E2abci[thread],v,tempt+j*v*v*o+i*v*v,v,0.0,Z[thread],v*v,thread);
      helper_->GPUTiledDGEMM_NoThread('n','t',v,v*v,o,-1.0,E2ijak+j*o*o*v+k*o*v,v,tempt+i*v*v*o,v*v,1.0,Z[thread],v,thread);

      //(ab)(ij)
      helper_->GPUTiledDGEMM_NoThread('t','t',v*v,v,v,1.0,E2abci[thread],v,tempt+i*v*v*o+j*v*v,v,0.0,Z2[thread],v*v,thread);
      helper_->GPUTiledDGEMM_NoThread('n','t',v,v*v,o,-1.0,E2ijak+i*o*o*v+k*o*v,v,tempt+j*v*v*o,v*v,1.0,Z2[thread],v,thread);
      for (int a=0; a<v; a++){
          for (int b=0; b<v; b++){
              C_DAXPY(v,1.0,Z2[thread]+b*v*v+a*v,1,Z[thread]+a*v*v+b*v,1);
          }
      }

      //(bc)(jk)
      addr = psio_get_address(PSIO_ZERO,(long int)j*v*v*v*sizeof(double));
      mypsio->read(PSIF_DCC_ABCI,"E2abci",(char*)&E2abci[thread][0],v*v*v*sizeof(double),addr,&addr);
      helper_->GPUTiledDGEMM_NoThread('t','t',v*v,v,v,1.0,E2abci[thread],v,tempt+k*v*v*o+i*v*v,v,0.0,Z2[thread],v*v,thread);
      helper_->GPUTiledDGEMM_NoThread('n','t',v,v*v,o,-1.0,E2ijak+k*o*o*v+j*o*v,v,tempt+i*v*v*o,v*v,1.0,Z2[thread],v,thread);
      for (int a=0; a<v; a++){
          for (int b=0; b<v; b++){
              C_DAXPY(v,1.0,Z2[thread]+a*v*v+b,v,Z[thread]+a*v*v+b*v,1);
          }
      }

      //(ikj)(acb)
      helper_->GPUTiledDGEMM_NoThread('t','t',v*v,v,v,1.0,E2abci[thread],v,tempt+i*v*v*o+k*v*v,v,0.0,Z2[thread],v*v,thread);
      helper_->GPUTiledDGEMM_NoThread('n','t',v,v*v,o,-1.0,E2ijak+i*o*o*v+j*o*v,v,tempt+k*v*v*o,v*v,1.0,Z2[thread],v,thread);
      for (int a=0; a<v; a++){
          for (int b=0; b<v; b++){
              C_DAXPY(v,1.0,Z2[thread]+a*v+b,v*v,Z[thread]+a*v*v+b*v,1);
          }
      }

      //(ac)(ik)
      addr = psio_get_address(PSIO_ZERO,(long int)i*v*v*v*sizeof(double));
      mypsio->read(PSIF_DCC_ABCI,"E2abci",(char*)&E2abci[thread][0],v*v*v*sizeof(double),addr,&addr);
      helper_->GPUTiledDGEMM_NoThread('t','t',v*v,v,v,1.0,E2abci[thread],v,tempt+j*v*v*o+k*v*v,v,0.0,Z2[thread],v*v,thread);
      helper_->GPUTiledDGEMM_NoThread('n','t',v,v*v,o,-1.0,E2ijak+j*o*o*v+i*o*v,v,tempt+k*v*v*o,v*v,1.0,Z2[thread],v,thread);
      for (int a=0; a<v; a++){
          for (int b=0; b<v; b++){
              C_DAXPY(v,1.0,Z2[thread]+b*v+a,v*v,Z[thread]+a*v*v+b*v,1);
          }
      }

      //(ijk)(abc)
      helper_->GPUTiledDGEMM_NoThread('t','t',v*v,v,v,1.0,E2abci[thread],v,tempt+k*v*v*o+j*v*v,v,0.0,Z2[thread],v*v,thread);
      helper_->GPUTiledDGEMM_NoThread('n','t',v,v*v,o,-1.0,E2ijak+k*o*o*v+i*o*v,v,tempt+j*v*v*o,v*v,1.0,Z2[thread],v,thread);
      for (int a=0; a<v; a++){
          for (int b=0; b<v; b++){
              C_DAXPY(v,1.0,Z2[thread]+b*v*v+a,v,Z[thread]+a*v*v+b*v,1);
          }
      }

      C_DCOPY(v*v*v,Z[thread],1,Z2[thread],1);
      for (int a=0; a<v; a++){
          double tai = t1[a*o+i];
          for (int b=0; b<v; b++){
              int ab = 1+(a==b);
              double tbj = t1[b*o+j];
              double E2iajb = E2klcd[i*v*v*o+a*v*o+j*v+b];
              for (int c=0; c<v; c++){
                  Z2[thread][a*v*v+b*v+c] += fac*(tai      *E2klcd[j*v*v*o+b*v*o+k*v+c] +
                                              tbj      *E2klcd[i*v*v*o+a*v*o+k*v+c] +
                                              t1[c*o+k]*E2iajb);
                  Z2[thread][a*v*v+b*v+c] /= (ab + (b==c) + (a==c));
              }
          }
      }

      for (int a=0; a<v; a++){
          for (int b=0; b<v; b++){
              for (int c=0; c<v; c++){
                  long int abc = a*v*v+b*v+c;
                  long int bac = b*v*v+a*v+c;
                  long int acb = a*v*v+c*v+b;
                  long int cba = c*v*v+b*v+a;

                  E2abci[thread][abc] = Z2[thread][acb] + Z2[thread][bac] + Z2[thread][cba];
              }
          }
      }
      double dijk = F[i]+F[j]+F[k];
      int ijkfac = ( 2-((i==j)+(j==k)+(i==k)) );
      // separate out these bits to save v^3 storage
      double tripval = 0.0;
      for (int a=0; a<v; a++){
          double dijka = dijk-F[a+o];
          for (int b=0; b<=a; b++){
              double dijkab = dijka-F[b+o];
              for (int c=0; c<=b; c++){
                  long int abc = a*v*v+b*v+c;
                  long int bca = b*v*v+c*v+a;
                  long int cab = c*v*v+a*v+b;
                  long int acb = a*v*v+c*v+b;
                  long int bac = b*v*v+a*v+c;
                  long int cba = c*v*v+b*v+a;
                  double dum      = Z[thread][abc]*Z2[thread][abc] + Z[thread][acb]*Z2[thread][acb]
                                  + Z[thread][bac]*Z2[thread][bac] + Z[thread][bca]*Z2[thread][bca]
                                  + Z[thread][cab]*Z2[thread][cab] + Z[thread][cba]*Z2[thread][cba];

                  dum            =  (E2abci[thread][abc])
                                 * ((Z[thread][abc] + Z[thread][bca] + Z[thread][cab])*-2.0
                                 +  (Z[thread][acb] + Z[thread][bac] + Z[thread][cba]))
                                 + 3.0*dum;
                  double denom = dijkab-F[c+o];
                  tripval += dum/denom;
              }
          }
      }
      etrip[thread] += tripval*ijkfac;
      // the second bit
      for (int a=0; a<v; a++){
          for (int b=0; b<v; b++){
              for (int c=0; c<v; c++){
                  long int abc = a*v*v+b*v+c;
                  long int bca = b*v*v+c*v+a;
                  long int cab = c*v*v+a*v+b;

                  E2abci[thread][abc]  = Z2[thread][abc] + Z2[thread][bca] + Z2[thread][cab];
              }
          }
      }
      tripval = 0.0;
      for (int a=0; a<v; a++){
          double dijka = dijk-F[a+o];
          for (int b=0; b<=a; b++){
              double dijkab = dijka-F[b+o];
              for (int c=0; c<=b; c++){
                  long int abc = a*v*v+b*v+c;
                  long int bca = b*v*v+c*v+a;
                  long int cab = c*v*v+a*v+b;
                  long int acb = a*v*v+c*v+b;
                  long int bac = b*v*v+a*v+c;
                  long int cba = c*v*v+b*v+a;

                  double dum     = (E2abci[thread][abc])
                                 * (Z[thread][abc] + Z[thread][bca] + Z[thread][cab]
                                 + (Z[thread][acb] + Z[thread][bac] + Z[thread][cba])*-2.0);

                  double denom = dijkab-F[c+o];
                  tripval += dum/denom;
              }
          }
      }
      etrip[thread] += tripval*ijkfac;
      // print out update 
      if (thread==0){
         int print = 0;
         stop = time(NULL);
         if ((double)ind/nijk >= 0.01 && !pct01){      pct01 = 1; print=1;}
         else if ((double)ind/nijk >= 0.02 && !pct02){ pct02 = 1; print=1;}
         else if ((double)ind/nijk >= 0.03 && !pct03){ pct03 = 1; print=1;}
         else if ((double)ind/nijk >= 0.04 && !pct04){ pct04 = 1; print=1;}
         else if ((double)ind/nijk >= 0.05 && !pct05){ pct05 = 1; print=1;}
         else if ((double)ind/nijk >= 0.1  && !pct10){ pct10 = 1; print=1;}
         else if ((double)ind/nijk >= 0.2  && !pct20){ pct20 = 1; print=1;}
         else if ((double)ind/nijk >= 0.3  && !pct30){ pct30 = 1; print=1;}
         else if ((double)ind/nijk >= 0.4  && !pct40){ pct40 = 1; print=1;}
         else if ((double)ind/nijk >= 0.5  && !pct50){ pct50 = 1; print=1;}
         else if ((double)ind/nijk >= 0.6  && !pct60){ pct60 = 1; print=1;}
         else if ((double)ind/nijk >= 0.7  && !pct70){ pct70 = 1; print=1;}
         else if ((double)ind/nijk >= 0.8  && !pct80){ pct80 = 1; print=1;}
         else if ((double)ind/nijk >= 0.9  && !pct90){ pct90 = 1; print=1;}
         if (print){
            outfile->Printf("              %3.1lf  %8d s\n",100.0*ind/nijk,(int)stop-(int)start);
            //fflush(outfile);
         }
      }
      mypsio->close(PSIF_DCC_ABCI,1);
      mypsio.reset();
  }

  double myet = 0.0;
  for (int i=0; i<nthreads; i++) myet += etrip[i];

  // ccsd(t) or qcisd(t)
  if (ccmethod <= 1) {
      et = myet;
      outfile->Printf("\n");
      outfile->Printf("        (T) energy   %s                   %20.12lf\n",space,et);
      outfile->Printf("\n");
      outfile->Printf("        %s(T) correlation energy       %20.12lf\n",name,eccsd+et);
      outfile->Printf("      * %s(T) total energy             %20.12lf\n",name,eccsd+et+escf);
      outfile->Printf("\n");
  }else {
      emp4_t = myet;
      outfile->Printf("\n");
      outfile->Printf("        MP4(T) correlation energy:         %20.12lf\n",emp4_t);
      outfile->Printf("\n");
      outfile->Printf("        MP4(SDTQ) correlation energy:      %20.12lf\n",emp2+emp3+emp4_sd+emp4_q+emp4_t);
      outfile->Printf("      * MP4(SDTQ) total energy:            %20.12lf\n",emp2+emp3+emp4_sd+emp4_q+emp4_t+escf);
      outfile->Printf("\n");
  }
  //fflush(outfile);

  // free memory:
  free(E2ijak);
  free(tempt);
  for (int i=0; i<nthreads; i++){  
      free(E2abci[i]);
      free(Z[i]);
      free(Z2[i]);
  }
  free(Z);
  free(Z2);
  free(E2abci);
  free(etrip);
  delete name;
  delete space;
            
  return Success;
}

}} // end of namespaces



